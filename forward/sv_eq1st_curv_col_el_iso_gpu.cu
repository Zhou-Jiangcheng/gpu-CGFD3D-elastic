#include "hip/hip_runtime.h"
/*******************************************************************************
 * solver of isotropic elastic 1st-order eqn using curv grid and collocated scheme
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <mpi.h>

#include "fdlib_mem.h"
#include "fdlib_math.h"
#include "sv_eq1st_curv_col_el_iso_gpu.h"
#include "cuda_common.h"

//#define SV_EQ1ST_CURV_COLGRD_ISO_DEBUG

/*******************************************************************************
 * perform one stage calculation of rhs
 ******************************************************************************/

void
sv_eq1st_curv_col_el_iso_onestage(
  float *w_cur_d,
  float *rhs_d, 
  wav_t  wav_d,
  fd_wav_t fd_wav_d,
  gdinfo_t  gdinfo_d,
  gdcurv_metric_t metric_d,
  md_t md_d,
  bdryfree_t bdryfree_d,
  bdrypml_t  bdrypml_d,
  src_t src_d,
  // include different order/stentil
  int num_of_fdx_op, fd_op_t *fdx_op,
  int num_of_fdy_op, fd_op_t *fdy_op,
  int num_of_fdz_op, fd_op_t *fdz_op,
  int fdz_max_len, 
  const int myid, const int verbose)
{
  // local pointer get each vars
  float *Vx    = w_cur_d + wav_d.Vx_pos ;
  float *Vy    = w_cur_d + wav_d.Vy_pos ;
  float *Vz    = w_cur_d + wav_d.Vz_pos ;
  float *Txx   = w_cur_d + wav_d.Txx_pos;
  float *Tyy   = w_cur_d + wav_d.Tyy_pos;
  float *Tzz   = w_cur_d + wav_d.Tzz_pos;
  float *Txz   = w_cur_d + wav_d.Txz_pos;
  float *Tyz   = w_cur_d + wav_d.Tyz_pos;
  float *Txy   = w_cur_d + wav_d.Txy_pos;
  float *hVx   = rhs_d   + wav_d.Vx_pos ; 
  float *hVy   = rhs_d   + wav_d.Vy_pos ; 
  float *hVz   = rhs_d   + wav_d.Vz_pos ; 
  float *hTxx  = rhs_d   + wav_d.Txx_pos; 
  float *hTyy  = rhs_d   + wav_d.Tyy_pos; 
  float *hTzz  = rhs_d   + wav_d.Tzz_pos; 
  float *hTxz  = rhs_d   + wav_d.Txz_pos; 
  float *hTyz  = rhs_d   + wav_d.Tyz_pos; 
  float *hTxy  = rhs_d   + wav_d.Txy_pos; 

  float *xi_x  = metric_d.xi_x;
  float *xi_y  = metric_d.xi_y;
  float *xi_z  = metric_d.xi_z;
  float *et_x  = metric_d.eta_x;
  float *et_y  = metric_d.eta_y;
  float *et_z  = metric_d.eta_z;
  float *zt_x  = metric_d.zeta_x;
  float *zt_y  = metric_d.zeta_y;
  float *zt_z  = metric_d.zeta_z;
  float *jac3d = metric_d.jac;

  float *lam3d = md_d.lambda;
  float * mu3d = md_d.mu;
  float *slw3d = md_d.rho;

  // grid size
  int ni1 = gdinfo_d.ni1;
  int ni2 = gdinfo_d.ni2;
  int nj1 = gdinfo_d.nj1;
  int nj2 = gdinfo_d.nj2;
  int nk1 = gdinfo_d.nk1;
  int nk2 = gdinfo_d.nk2;

  int ni  = gdinfo_d.ni;
  int nj  = gdinfo_d.nj;
  int nk  = gdinfo_d.nk;
  int nx  = gdinfo_d.nx;
  int ny  = gdinfo_d.ny;
  int nz  = gdinfo_d.nz;
  size_t siz_line   = gdinfo_d.siz_line;
  size_t siz_slice  = gdinfo_d.siz_slice;
  size_t siz_volume = gdinfo_d.siz_volume;

  float *matVx2Vz = bdryfree_d.matVx2Vz2;
  float *matVy2Vz = bdryfree_d.matVy2Vz2;

  // local fd op
  int    fdx_len;
  int    *fdx_indx;
  float  *fdx_coef;
  int    fdy_len;
  int    *fdy_indx;
  float  *fdy_coef;
  int    fdz_len;
  int    *fdz_indx;
  float  *fdz_coef;

  // for get a op from 1d array, currently use num_of_fdz_op as index
  // length, index, coef of a op
  fdx_len  = fdx_op[num_of_fdx_op-1].total_len;
  fdx_indx = fdx_op[num_of_fdx_op-1].indx;
  fdx_coef = fdx_op[num_of_fdx_op-1].coef;

  fdy_len  = fdy_op[num_of_fdy_op-1].total_len;
  fdy_indx = fdy_op[num_of_fdy_op-1].indx;
  fdy_coef = fdy_op[num_of_fdy_op-1].coef;

  fdz_len  = fdz_op[num_of_fdz_op-1].total_len;
  fdz_indx = fdz_op[num_of_fdz_op-1].indx;
  fdz_coef = fdz_op[num_of_fdz_op-1].coef;

  // use local stack array for speedup
  float  lfdx_coef [fdx_len];
  size_t lfdx_shift[fdx_len];
  float  lfdy_coef [fdy_len];
  size_t lfdy_shift[fdy_len];
  float  lfdz_coef [fdz_len];
  size_t lfdz_shift[fdz_len];

  // put fd op into local array
  for (int i=0; i < fdx_len; i++) {
    lfdx_coef [i] = fdx_coef[i];
    lfdx_shift[i] = fdx_indx[i];
  }
  for (int j=0; j < fdy_len; j++) {
    lfdy_coef [j] = fdy_coef[j];
    lfdy_shift[j] = fdy_indx[j] * siz_line;
  }
  for (int k=0; k < fdz_len; k++) {
    lfdz_coef [k] = fdz_coef[k];
    lfdz_shift[k] = fdz_indx[k] * siz_slice;
  }

  // allocate max_len because fdz may have different lens
  // these array is for low order surface
  float  fdz_coef_all [num_of_fdz_op*fdz_max_len];
  size_t fdz_shift_all[num_of_fdz_op*fdz_max_len];
  int    fdz_len_all[num_of_fdz_op];
  // loop near surface layers
  for (int n=0; n < num_of_fdz_op-1; n++)
  {
    // get pos and len for this point
    fdz_len_all[n]  = fdz_op[n].total_len;
    // point to indx/coef for this point
    int   *p_fdz_indx  = fdz_op[n].indx;
    float *p_fdz_coef  = fdz_op[n].coef;
    for (int n_fd = 0; n_fd < fdz_len_all[n] ; n_fd++) {
      fdz_shift_all[n_fd + n*fdz_max_len]  = p_fdz_indx[n_fd] * siz_slice;
      fdz_coef_all [n_fd + n*fdz_max_len]  = p_fdz_coef[n_fd];
    }
  }

  int  *lfdz_len_d = fd_wav_d.fdz_len_d;
  float *lfdx_coef_d = fd_wav_d.fdx_coef_d;
  float *lfdy_coef_d = fd_wav_d.fdy_coef_d;
  float *lfdz_coef_d = fd_wav_d.fdz_coef_d;
  float *lfdz_coef_all_d = fd_wav_d.fdz_coef_all_d;
  size_t  *lfdx_shift_d = fd_wav_d.fdx_shift_d;
  size_t  *lfdy_shift_d = fd_wav_d.fdy_shift_d;
  size_t  *lfdz_shift_d = fd_wav_d.fdz_shift_d;
  size_t  *lfdz_shift_all_d = fd_wav_d.fdz_shift_all_d;
  int  *lfdx_indx_d = fd_wav_d.fdx_indx_d;
  int  *lfdy_indx_d = fd_wav_d.fdy_indx_d;
  int  *lfdz_indx_d = fd_wav_d.fdz_indx_d;
  int  *lfdz_indx_all_d = fd_wav_d.fdz_indx_all_d;
  //host to device
  CUDACHECK(hipMemcpy(lfdx_coef_d,lfdx_coef,fdx_len*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdy_coef_d,lfdy_coef,fdy_len*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_coef_d,lfdz_coef,fdz_len*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdx_shift_d,lfdx_shift,fdx_len*sizeof(size_t),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdy_shift_d,lfdy_shift,fdy_len*sizeof(size_t),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_shift_d,lfdz_shift,fdz_len*sizeof(size_t),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdx_indx_d,fdx_indx,fdx_len*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdy_indx_d,fdy_indx,fdy_len*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_indx_d,fdz_indx,fdz_len*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_len_d,fdz_len_all,num_of_fdz_op*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_coef_all_d,fdz_coef_all,fdz_max_len*num_of_fdz_op*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_shift_all_d,fdz_shift_all,fdz_max_len*num_of_fdz_op*sizeof(size_t),hipMemcpyHostToDevice));
  
  {
    dim3 block(8,8,8);
    dim3 grid;
    grid.x = (ni+block.x-1)/block.x;
    grid.y = (nj+block.y-1)/block.y;
    grid.z = (nk+block.z-1)/block.z;
    sv_eq1st_curv_col_el_iso_rhs_inner_gpu <<<grid, block>>> (
                        Vx,Vy,Vz,Txx,Tyy,Tzz,Txz,Tyz,Txy,
                        hVx,hVy,hVz,hTxx,hTyy,hTzz,hTxz,hTyz,hTxy,
                        xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                        lam3d, mu3d, slw3d,
                        ni1,ni,nj1,nj,nk1,nk,siz_line,siz_slice,
                        fdx_len, lfdx_shift_d, lfdx_coef_d,
                        fdy_len, lfdy_shift_d, lfdy_coef_d,
                        fdz_len, lfdz_shift_d, lfdz_coef_d,
                        myid, verbose);
    CUDACHECK( hipDeviceSynchronize() );
  }

  // free, abs, source in turn
  // free surface at z2
  if (bdryfree_d.is_at_sides[2][1] == 1)
  {
    // tractiong
    {
      dim3 block(8,8);
      dim3 grid;
      grid.x = (ni+block.x-1)/block.x;
      grid.y = (nj+block.y-1)/block.y;
      sv_eq1st_curv_col_el_iso_rhs_timg_z2_gpu  <<<grid, block>>> (
                          Txx,Tyy,Tzz,Txz,Tyz,Txy,hVx,hVy,hVz,
                          xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                          jac3d, slw3d,
                          ni1,ni,nj1,nj,nk1,nk2,siz_line,siz_slice,
                          fdx_len, lfdx_indx_d, lfdx_coef_d,
                          fdy_len, lfdy_indx_d, lfdy_coef_d,
                          fdz_len, lfdz_indx_d, lfdz_coef_d,
                          myid, verbose);
      hipDeviceSynchronize();
    }
    // velocity: vlow
    {
      dim3 block(8,8);
      dim3 grid;
      grid.x = (ni+block.x-1)/block.x;
      grid.y = (nj+block.y-1)/block.y;
      sv_eq1st_curv_col_el_iso_rhs_vlow_z2_gpu  <<<grid, block>>> (
                        Vx,Vy,Vz,hTxx,hTyy,hTzz,hTxz,hTyz,hTxy,
                        xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                        lam3d, mu3d, slw3d,
                        matVx2Vz,matVy2Vz,
                        ni1,ni,nj1,nj,nk1,nk2,siz_line,siz_slice,
                        fdx_len, lfdx_shift_d, lfdx_coef_d,
                        fdy_len, lfdy_shift_d, lfdy_coef_d,
                        num_of_fdz_op,fdz_max_len,lfdz_len_d,
                        lfdz_coef_all_d,lfdz_shift_all_d,
                        myid, verbose);
      CUDACHECK( hipDeviceSynchronize() );
    }
  }

  // cfs-pml, loop face inside
  if (bdrypml_d.is_enable == 1)
  {
    sv_eq1st_curv_col_el_iso_rhs_cfspml(Vx,Vy,Vz,Txx,Tyy,Tzz,Txz,Tyz,Txy,
                                        hVx,hVy,hVz,hTxx,hTyy,hTzz,hTxz,hTyz,hTxy,
                                        xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                                        lam3d, mu3d, slw3d,
                                        nk2, siz_line,siz_slice,
                                        fdx_len, lfdx_shift_d, lfdx_coef_d,
                                        fdy_len, lfdy_shift_d, lfdy_coef_d,
                                        fdz_len, lfdz_shift_d, lfdz_coef_d,
                                        bdrypml_d, bdryfree_d,
                                        myid, verbose);
  }

  // add source term
  if (src_d.total_number > 0)
  {
    {
      dim3 block(256);
      dim3 grid;
      grid.x = (src_d.total_number+block.x-1) / block.x;
      sv_eq1st_curv_col_el_iso_rhs_src_gpu  <<< grid,block >>> (
                        hVx, hVy, hVz, hTxx, hTyy, hTzz, hTxz, hTyz, hTxy,
                        jac3d, slw3d, 
                        src_d,
                        myid, verbose);
      CUDACHECK( hipDeviceSynchronize() );
    }
  }
  
  // end func
  return;
}

/*******************************************************************************
 * calculate all points without boundaries treatment
 ******************************************************************************/

__global__ void
sv_eq1st_curv_col_el_iso_rhs_inner_gpu(
    float *  Vx , float *  Vy , float *  Vz ,
    float *  Txx, float *  Tyy, float *  Tzz,
    float *  Txz, float *  Tyz, float *  Txy,
    float * hVx , float * hVy , float * hVz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float * lam3d, float * mu3d, float * slw3d,
    int ni1, int ni, int nj1, int nj, int nk1, int nk,
    size_t siz_line, size_t siz_slice,
    int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
    int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
    int fdz_len, size_t * lfdz_shift, float * lfdz_coef,
    const int myid, const int verbose)
{
  
  // local var
  float DxTxx,DxTyy,DxTzz,DxTxy,DxTxz,DxTyz,DxVx,DxVy,DxVz;
  float DyTxx,DyTyy,DyTzz,DyTxy,DyTxz,DyTyz,DyVx,DyVy,DyVz;
  float DzTxx,DzTyy,DzTzz,DzTxy,DzTxz,DzTyz,DzVx,DzVy,DzVz;
  float lam,mu,lam2mu,slw;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;

  float * Vx_ptr;
  float * Vy_ptr;
  float * Vz_ptr;
  float * Txx_ptr;
  float * Txy_ptr;
  float * Txz_ptr;
  float * Tyy_ptr;
  float * Tzz_ptr;
  float * Tyz_ptr;


  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  size_t iz = blockIdx.z * blockDim.z + threadIdx.z;

  // caclu all points
  if(ix<ni && iy<nj && iz<nk)
  {
    size_t iptr = (ix+ni1) + (iy+nj1) * siz_line + (iz+nk1) * siz_slice;

    Vx_ptr = Vx + iptr;
    Vy_ptr = Vy + iptr;
    Vz_ptr = Vz + iptr;
    Txx_ptr = Txx + iptr;
    Tyy_ptr = Tyy + iptr;
    Tzz_ptr = Tzz + iptr;
    Txz_ptr = Txz + iptr;
    Tyz_ptr = Tyz + iptr;
    Txy_ptr = Txy + iptr;

    // Vx derivatives
    M_FD_SHIFT_PTR_MACDRP(DxVx, Vx_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyVx, Vx_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzVx, Vx_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Vy derivatives
    M_FD_SHIFT_PTR_MACDRP(DxVy, Vy_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyVy, Vy_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzVy, Vy_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Vz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxVz, Vz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyVz, Vz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzVz, Vz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Txx derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTxx, Txx_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTxx, Txx_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTxx, Txx_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Tyy derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTyy, Tyy_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTyy, Tyy_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTyy, Tyy_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Tzz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTzz, Tzz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTzz, Tzz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTzz, Tzz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Txz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTxz, Txz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTxz, Txz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTxz, Txz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Tyz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTyz, Tyz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTyz, Tyz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTyz, Tyz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Txy derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTxy, Txy_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTxy, Txy_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTxy, Txy_ptr, fdz_len, lfdz_shift, lfdz_coef);

    
    // metric
    xix = xi_x[iptr];
    xiy = xi_y[iptr];
    xiz = xi_z[iptr];
    etx = et_x[iptr];
    ety = et_y[iptr];
    etz = et_z[iptr];
    ztx = zt_x[iptr];
    zty = zt_y[iptr];
    ztz = zt_z[iptr];

    // medium
    lam = lam3d[iptr];
    mu  =  mu3d[iptr];
    slw = slw3d[iptr];
    lam2mu = lam + 2.0 * mu;

    // moment equation
    hVx[iptr] = slw*( xix*DxTxx + xiy*DxTxy + xiz*DxTxz  
                     +etx*DyTxx + ety*DyTxy + etz*DyTxz 
                     +ztx*DzTxx + zty*DzTxy + ztz*DzTxz );
    hVy[iptr] = slw*( xix*DxTxy + xiy*DxTyy + xiz*DxTyz
                     +etx*DyTxy + ety*DyTyy + etz*DyTyz
                     +ztx*DzTxy + zty*DzTyy + ztz*DzTyz );
    hVz[iptr] = slw*( xix*DxTxz + xiy*DxTyz + xiz*DxTzz 
                     +etx*DyTxz + ety*DyTyz + etz*DyTzz
                     +ztx*DzTxz + zty*DzTyz + ztz*DzTzz );

    // Hooke's equatoin
    hTxx[iptr] =  lam2mu * ( xix*DxVx  +etx*DyVx + ztx*DzVx)
                + lam    * ( xiy*DxVy + ety*DyVy + zty*DzVy
                            +xiz*DxVz + etz*DyVz + ztz*DzVz);

    hTyy[iptr] = lam2mu * ( xiy*DxVy + ety*DyVy + zty*DzVy)
                +lam    * ( xix*DxVx + etx*DyVx + ztx*DzVx
                           +xiz*DxVz + etz*DyVz + ztz*DzVz);

    hTzz[iptr] = lam2mu * ( xiz*DxVz + etz*DyVz + ztz*DzVz)
                +lam    * ( xix*DxVx  +etx*DyVx  +ztx*DzVx
                           +xiy*DxVy + ety*DyVy + zty*DzVy);

    hTxy[iptr] = mu *(
                 xiy*DxVx + xix*DxVy
                +ety*DyVx + etx*DyVy
                +zty*DzVx + ztx*DzVy
                );
    hTxz[iptr] = mu *(
                 xiz*DxVx + xix*DxVz
                +etz*DyVx + etx*DyVz
                +ztz*DzVx + ztx*DzVz
                );
    hTyz[iptr] = mu *(
                 xiz*DxVy + xiy*DxVz
                +etz*DyVy + ety*DyVz
                +ztz*DzVy + zty*DzVz
                );
  }

  return;
}

/*******************************************************************************
 * free surface boundary
 ******************************************************************************/

/*
 * implement traction image boundary 
 */

__global__ void
sv_eq1st_curv_col_el_iso_rhs_timg_z2_gpu(
    float *  Txx, float *  Tyy, float *  Tzz,
    float *  Txz, float *  Tyz, float *  Txy,
    float * hVx , float * hVy , float * hVz ,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float * jac3d, float * slw3d,
    int ni1, int ni, int nj1, int nj, int nk1, int nk2,
    size_t siz_line, size_t siz_slice, 
    int fdx_len, int * fdx_indx, float * lfdx_coef,
    int fdy_len, int * fdy_indx, float * lfdy_coef,
    int fdz_len, int * fdz_indx, float * lfdz_coef,
    const int myid, const int verbose)
{

  // loop var for fd
  int n_fd; // loop var for fd

  // local var
  float DxTx,DyTy,DzTz;
  float slwjac;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;

  // to save traction and other two dir force var
  float vecxi[5] = {0.0};
  float vecet[5] = {0.0};
  float veczt[5] = {0.0};
  int n, iptr4vec;

  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;

  // last indx, free surface force Tx/Ty/Tz to 0 in cal
  size_t k_min = nk2 - fdz_indx[fdz_len-1];

  // point affected by timg
  for (size_t k=k_min; k <= nk2; k++)
  {
    // k corresponding to 0 index of the fd op

    // index of free surface
    int n_free = nk2 - k - fdz_indx[0]; // first indx is negative

    if(ix<ni && iy<nj)
    {

      size_t iptr = (ix+ni1) + (iy+nj1) * siz_line + k * siz_slice;
      // metric
      xix = xi_x[iptr];
      xiy = xi_y[iptr];
      xiz = xi_z[iptr];
      etx = et_x[iptr];
      ety = et_y[iptr];
      etz = et_z[iptr];
      ztx = zt_x[iptr];
      zty = zt_y[iptr];
      ztz = zt_z[iptr];

      // slowness and jac
      slwjac = slw3d[iptr] / jac3d[iptr];

      //
      // for hVx
      //

      // transform to conservative vars
      for (n=0; n<fdx_len; n++) {
        iptr4vec = iptr + fdx_indx[n];
        vecxi[n] = jac3d[iptr4vec] * (  xi_x[iptr4vec] * Txx[iptr4vec]
                                      + xi_y[iptr4vec] * Txy[iptr4vec]
                                      + xi_z[iptr4vec] * Txz[iptr4vec] );
      }
      for (n=0; n<fdy_len; n++) {
        iptr4vec = iptr + fdy_indx[n] * siz_line;
        vecet[n] = jac3d[iptr4vec] * (  et_x[iptr4vec] * Txx[iptr4vec]
                                      + et_y[iptr4vec] * Txy[iptr4vec]
                                      + et_z[iptr4vec] * Txz[iptr4vec] );
      }

      // blow surface -> cal
      for (n=0; n<n_free; n++) {
        iptr4vec = iptr + fdz_indx[n] * siz_slice;
        veczt[n] = jac3d[iptr4vec] * (  zt_x[iptr4vec] * Txx[iptr4vec]
                                      + zt_y[iptr4vec] * Txy[iptr4vec]
                                      + zt_z[iptr4vec] * Txz[iptr4vec] );
      }

      // at surface -> set to 0
      veczt[n_free] = 0.0;

      // above surface -> mirror
      for (n=n_free+1; n<fdz_len; n++)
      {
        int n_img = fdz_indx[n] - 2*(n-n_free);
        iptr4vec = iptr + n_img * siz_slice;
        veczt[n] = -jac3d[iptr4vec] * (  zt_x[iptr4vec] * Txx[iptr4vec]
                                       + zt_y[iptr4vec] * Txy[iptr4vec]
                                       + zt_z[iptr4vec] * Txz[iptr4vec] );
      }

      // deri
      M_FD_NOINDX(DxTx, vecxi, fdx_len, lfdx_coef, n_fd);
      M_FD_NOINDX(DyTy, vecet, fdy_len, lfdy_coef, n_fd);
      M_FD_NOINDX(DzTz, veczt, fdz_len, lfdz_coef, n_fd);

      hVx[iptr] = ( DxTx+DyTy+DzTz ) * slwjac;

      //
      // for hVy
      //

      // transform to conservative vars
      for (n=0; n<fdx_len; n++) {
        iptr4vec = iptr + fdx_indx[n];
        vecxi[n] = jac3d[iptr4vec] * (  xi_x[iptr4vec] * Txy[iptr4vec]
                                      + xi_y[iptr4vec] * Tyy[iptr4vec]
                                      + xi_z[iptr4vec] * Tyz[iptr4vec] );
      }
      for (n=0; n<fdy_len; n++) {
        iptr4vec = iptr + fdy_indx[n] * siz_line;
        vecet[n] = jac3d[iptr4vec] * (  et_x[iptr4vec] * Txy[iptr4vec]
                                      + et_y[iptr4vec] * Tyy[iptr4vec]
                                      + et_z[iptr4vec] * Tyz[iptr4vec] );
      }

      // blow surface -> cal
      for (n=0; n<n_free; n++) {
        iptr4vec = iptr + fdz_indx[n] * siz_slice;
        veczt[n] = jac3d[iptr4vec] * (  zt_x[iptr4vec] * Txy[iptr4vec]
                                      + zt_y[iptr4vec] * Tyy[iptr4vec]
                                      + zt_z[iptr4vec] * Tyz[iptr4vec] );
      }

      // at surface -> set to 0
      veczt[n_free] = 0.0;

      // above surface -> mirror
      for (n=n_free+1; n<fdz_len; n++) {
        int n_img = fdz_indx[n] - 2*(n-n_free);
        iptr4vec = iptr + n_img * siz_slice;
        veczt[n] = -jac3d[iptr4vec] * (  zt_x[iptr4vec] * Txy[iptr4vec]
                                       + zt_y[iptr4vec] * Tyy[iptr4vec]
                                       + zt_z[iptr4vec] * Tyz[iptr4vec] );
      }

      // deri
      M_FD_NOINDX(DxTx, vecxi, fdx_len, lfdx_coef, n_fd);
      M_FD_NOINDX(DyTy, vecet, fdy_len, lfdy_coef, n_fd);
      M_FD_NOINDX(DzTz, veczt, fdz_len, lfdz_coef, n_fd);

      hVy[iptr] = ( DxTx+DyTy+DzTz ) * slwjac;

      //
      // for hVz
      //

      // transform to conservative vars
      for (n=0; n<fdx_len; n++) {
        iptr4vec = iptr + fdx_indx[n];
        vecxi[n] = jac3d[iptr4vec] * (  xi_x[iptr4vec] * Txz[iptr4vec]
                                      + xi_y[iptr4vec] * Tyz[iptr4vec]
                                      + xi_z[iptr4vec] * Tzz[iptr4vec] );
      }
      for (n=0; n<fdy_len; n++) {
        iptr4vec = iptr + fdy_indx[n] * siz_line;
        vecet[n] = jac3d[iptr4vec] * (  et_x[iptr4vec] * Txz[iptr4vec]
                                      + et_y[iptr4vec] * Tyz[iptr4vec]
                                      + et_z[iptr4vec] * Tzz[iptr4vec] );
      }

      // blow surface -> cal
      for (n=0; n<n_free; n++) {
        iptr4vec = iptr + fdz_indx[n] * siz_slice;
        veczt[n] = jac3d[iptr4vec] * (  zt_x[iptr4vec] * Txz[iptr4vec]
                                      + zt_y[iptr4vec] * Tyz[iptr4vec]
                                      + zt_z[iptr4vec] * Tzz[iptr4vec] );
      }

      // at surface -> set to 0
      veczt[n_free] = 0.0;

      // above surface -> mirror
      for (n=n_free+1; n<fdz_len; n++) {
        int n_img = fdz_indx[n] - 2*(n-n_free);
        iptr4vec = iptr + n_img * siz_slice;
        veczt[n] = -jac3d[iptr4vec] * (  zt_x[iptr4vec] * Txz[iptr4vec]
                                       + zt_y[iptr4vec] * Tyz[iptr4vec]
                                       + zt_z[iptr4vec] * Tzz[iptr4vec] );
      }

      // for hVx 
      M_FD_NOINDX(DxTx, vecxi, fdx_len, lfdx_coef, n_fd);
      M_FD_NOINDX(DyTy, vecet, fdy_len, lfdy_coef, n_fd);
      M_FD_NOINDX(DzTz, veczt, fdz_len, lfdz_coef, n_fd);

      hVz[iptr] = ( DxTx+DyTy+DzTz ) * slwjac;
    }
  }

  return;
}

/*
 * implement vlow boundary
 */

__global__ void
sv_eq1st_curv_col_el_iso_rhs_vlow_z2_gpu(
    float *  Vx , float *  Vy , float *  Vz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float * lam3d, float * mu3d, float * slw3d,
    float * matVx2Vz, float * matVy2Vz,
    int ni1, int ni, int nj1, int nj, int nk1, int nk2,
    size_t siz_line, size_t siz_slice,
    int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
    int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
    int num_of_fdz_op, int fdz_max_len, int * fdz_len,
    float *lfdz_coef_all, size_t *lfdz_shift_all,
    const int myid, const int verbose)
{

  // local var
  int k;
  int n_fd; // loop var for fd
  int lfdz_len;
  // local var
  float DxVx,DxVy,DxVz;
  float DyVx,DyVy,DyVz;
  float DzVx,DzVy,DzVz;
  float lam,mu,lam2mu,slw;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;

  float lfdz_coef[5] = {0.0};
  int   lfdz_shift[5] = {0};
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  // loop near surface layers
  for (int n=0; n < num_of_fdz_op-1; n++)
  {
    // conver to k index, from surface to inner
    k = nk2 - n;
    // get pos and len for this point
    lfdz_len  = fdz_len[n];
    for (n_fd = 0; n_fd < lfdz_len ; n_fd++) {
      lfdz_shift[n_fd] = lfdz_shift_all[n*fdz_max_len+n_fd];
      lfdz_coef [n_fd]  = lfdz_coef_all [n*fdz_max_len+n_fd];
    }

    if(ix<ni && iy<nj)
    {
      size_t iptr   = (ix+ni1) + (iy+nj1) * siz_line + k * siz_slice;

      // metric
      xix = xi_x[iptr];
      xiy = xi_y[iptr];
      xiz = xi_z[iptr];
      etx = et_x[iptr];
      ety = et_y[iptr];
      etz = et_z[iptr];
      ztx = zt_x[iptr];
      zty = zt_y[iptr];
      ztz = zt_z[iptr];

      // medium
      lam = lam3d[iptr];
      mu  =  mu3d[iptr];
      slw = slw3d[iptr];
      lam2mu = lam + 2.0 * mu;

      // Vx derivatives
      M_FD_SHIFT(DxVx, Vx, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DyVx, Vx, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      // Vy derivatives
      M_FD_SHIFT(DxVy, Vy, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DyVy, Vy, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      // Vz derivatives
      M_FD_SHIFT(DxVz, Vz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DyVz, Vz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      if (k==nk2) // at surface, convert
      {
        size_t ij = ((ix+ni1) + (iy+nj1) * siz_line)*9;
        DzVx = matVx2Vz[ij+3*0+0] * DxVx
             + matVx2Vz[ij+3*0+1] * DxVy
             + matVx2Vz[ij+3*0+2] * DxVz
             + matVy2Vz[ij+3*0+0] * DyVx
             + matVy2Vz[ij+3*0+1] * DyVy
             + matVy2Vz[ij+3*0+2] * DyVz;

        DzVy = matVx2Vz[ij+3*1+0] * DxVx
             + matVx2Vz[ij+3*1+1] * DxVy
             + matVx2Vz[ij+3*1+2] * DxVz
             + matVy2Vz[ij+3*1+0] * DyVx
             + matVy2Vz[ij+3*1+1] * DyVy
             + matVy2Vz[ij+3*1+2] * DyVz;

        DzVz = matVx2Vz[ij+3*2+0] * DxVx
             + matVx2Vz[ij+3*2+1] * DxVy
             + matVx2Vz[ij+3*2+2] * DxVz
             + matVy2Vz[ij+3*2+0] * DyVx
             + matVy2Vz[ij+3*2+1] * DyVy
             + matVy2Vz[ij+3*2+2] * DyVz;
      }
      else // lower than surface, lower order
      {
        M_FD_SHIFT(DzVx, Vx, iptr, lfdz_len, lfdz_shift, lfdz_coef, n_fd);
        M_FD_SHIFT(DzVy, Vy, iptr, lfdz_len, lfdz_shift, lfdz_coef, n_fd);
        M_FD_SHIFT(DzVz, Vz, iptr, lfdz_len, lfdz_shift, lfdz_coef, n_fd);
      }

      // Hooke's equatoin
      hTxx[iptr] =  lam2mu * ( xix*DxVx  +etx*DyVx + ztx*DzVx)
                  + lam    * ( xiy*DxVy + ety*DyVy + zty*DzVy
                              +xiz*DxVz + etz*DyVz + ztz*DzVz);

      hTyy[iptr] = lam2mu * ( xiy*DxVy + ety*DyVy + zty*DzVy)
                  +lam    * ( xix*DxVx + etx*DyVx + ztx*DzVx
                             +xiz*DxVz + etz*DyVz + ztz*DzVz);

      hTzz[iptr] = lam2mu * ( xiz*DxVz + etz*DyVz + ztz*DzVz)
                  +lam    * ( xix*DxVx  +etx*DyVx  +ztx*DzVx
                             +xiy*DxVy + ety*DyVy + zty*DzVy);

      hTxy[iptr] = mu *(
                   xiy*DxVx + xix*DxVy
                  +ety*DyVx + etx*DyVy
                  +zty*DzVx + ztx*DzVy
                  );
      hTxz[iptr] = mu *(
                   xiz*DxVx + xix*DxVz
                  +etz*DyVx + etx*DyVz
                  +ztz*DzVx + ztx*DzVz
                  );
      hTyz[iptr] = mu *(
                   xiz*DxVy + xiy*DxVz
                  +etz*DyVy + ety*DyVz
                  +ztz*DzVy + zty*DzVz
                  );
    }
  }

  return;
}

/*******************************************************************************
 * CFS-PML boundary
 ******************************************************************************/

/*
 * cfspml, reference to each pml var inside function
 */

void
sv_eq1st_curv_col_el_iso_rhs_cfspml(
    float *  Vx , float *  Vy , float *  Vz ,
    float *  Txx, float *  Tyy, float *  Tzz,
    float *  Txz, float *  Tyz, float *  Txy,
    float * hVx , float * hVy , float * hVz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float * lam3d, float *  mu3d, float * slw3d,
    int nk2, size_t siz_line, size_t siz_slice,
    int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
    int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
    int fdz_len, size_t * lfdz_shift, float * lfdz_coef,
    bdrypml_t bdrypml, bdryfree_t bdryfree,
    const int myid, const int verbose)
{
  // check each side
  for (int idim=0; idim<CONST_NDIM; idim++)
  {
    for (int iside=0; iside<2; iside++)
    {
      // skip to next face if not cfspml
      if (bdrypml.is_at_sides[idim][iside] == 0) continue;

      // get index into local var
      int abs_ni1 = bdrypml.ni1[idim][iside];
      int abs_ni2 = bdrypml.ni2[idim][iside];
      int abs_nj1 = bdrypml.nj1[idim][iside];
      int abs_nj2 = bdrypml.nj2[idim][iside];
      int abs_nk1 = bdrypml.nk1[idim][iside];
      int abs_nk2 = bdrypml.nk2[idim][iside];

      
      int abs_ni = abs_ni2-abs_ni1+1; 
      int abs_nj = abs_nj2-abs_nj1+1; 
      int abs_nk = abs_nk2-abs_nk1+1; 
      {
        dim3 block(8,4,4);
        dim3 grid;
        grid.x = (abs_ni+block.x-1)/block.x;
        grid.y = (abs_nj+block.y-1)/block.y;
        grid.z = (abs_nk+block.z-1)/block.z;

        sv_eq1st_curv_col_el_iso_rhs_cfspml_gpu <<<grid, block>>> (
                                idim, iside,
                                Vx , Vy , Vz , Txx,  Tyy,  Tzz,
                                Txz,  Tyz,  Txy, hVx , hVy , hVz,
                                hTxx, hTyy, hTzz, hTxz, hTyz, hTxy,
                                xi_x, xi_y, xi_z, et_x, et_y, et_z,
                                zt_x, zt_y, zt_z, lam3d, mu3d, slw3d,
                                nk2, siz_line, siz_slice,
                                fdx_len, lfdx_shift,  lfdx_coef,
                                fdy_len, lfdy_shift,  lfdy_coef,
                                fdz_len, lfdz_shift,  lfdz_coef,
                                bdrypml, bdryfree, myid, verbose);
        //hipDeviceSynchronize();
      }
    } // iside
  } // idim

  return;
}

__global__ void
sv_eq1st_curv_col_el_iso_rhs_cfspml_gpu(int idim, int iside,
                                        float *  Vx , float *  Vy , float *  Vz ,
                                        float *  Txx, float *  Tyy, float *  Tzz,
                                        float *  Txz, float *  Tyz, float *  Txy,
                                        float * hVx , float * hVy , float * hVz ,
                                        float * hTxx, float * hTyy, float * hTzz,
                                        float * hTxz, float * hTyz, float * hTxy,
                                        float * xi_x, float * xi_y, float * xi_z,
                                        float * et_x, float * et_y, float * et_z,
                                        float * zt_x, float * zt_y, float * zt_z,
                                        float * lam3d, float *  mu3d, float * slw3d,
                                        int nk2, size_t siz_line, size_t siz_slice,
                                        int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
                                        int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
                                        int fdz_len, size_t * lfdz_shift, float * lfdz_coef,
                                        bdrypml_t bdrypml, bdryfree_t bdryfree,
                                        const int myid, const int verbose)
{
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  size_t iz = blockIdx.z * blockDim.z + threadIdx.z;
  float *matVx2Vz = bdryfree.matVx2Vz2;
  float *matVy2Vz = bdryfree.matVy2Vz2;
  // local
  size_t iptr, iptr_a;
  float coef_A, coef_B, coef_D, coef_B_minus_1;
  // loop var for fd
  int n_fd;

  // get index into local var
  int abs_ni1 = bdrypml.ni1[idim][iside];
  int abs_ni2 = bdrypml.ni2[idim][iside];
  int abs_nj1 = bdrypml.nj1[idim][iside];
  int abs_nj2 = bdrypml.nj2[idim][iside];
  int abs_nk1 = bdrypml.nk1[idim][iside];
  int abs_nk2 = bdrypml.nk2[idim][iside];

  
  int abs_ni = abs_ni2-abs_ni1+1; 
  int abs_nj = abs_nj2-abs_nj1+1; 
  int abs_nk = abs_nk2-abs_nk1+1; 

  // val on point
  float DxTxx,DxTyy,DxTzz,DxTxy,DxTxz,DxTyz,DxVx,DxVy,DxVz;
  float DyTxx,DyTyy,DyTzz,DyTxy,DyTxz,DyTyz,DyVx,DyVy,DyVz;
  float DzTxx,DzTyy,DzTzz,DzTxy,DzTxz,DzTyz,DzVx,DzVy,DzVz;
  float lam,mu,lam2mu,slw;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;
  float hVx_rhs,hVy_rhs,hVz_rhs;
  float hTxx_rhs,hTyy_rhs,hTzz_rhs,hTxz_rhs,hTyz_rhs,hTxy_rhs;
  // for free surface
  float Dx_DzVx,Dy_DzVx,Dx_DzVy,Dy_DzVy,Dx_DzVz,Dy_DzVz;
  // get coef for this face
  float * ptr_coef_A = bdrypml.A[idim][iside];
  float * ptr_coef_B = bdrypml.B[idim][iside];
  float * ptr_coef_D = bdrypml.D[idim][iside];

  bdrypml_auxvar_t *auxvar = &(bdrypml.auxvar[idim][iside]);

  // get pml vars
  float * abs_vars_cur = auxvar->cur;
  float * abs_vars_rhs = auxvar->rhs;

  float * pml_Vx   = abs_vars_cur + auxvar->Vx_pos;
  float * pml_Vy   = abs_vars_cur + auxvar->Vy_pos;
  float * pml_Vz   = abs_vars_cur + auxvar->Vz_pos;
  float * pml_Txx  = abs_vars_cur + auxvar->Txx_pos;
  float * pml_Tyy  = abs_vars_cur + auxvar->Tyy_pos;
  float * pml_Tzz  = abs_vars_cur + auxvar->Tzz_pos;
  float * pml_Txz  = abs_vars_cur + auxvar->Txz_pos;
  float * pml_Tyz  = abs_vars_cur + auxvar->Tyz_pos;
  float * pml_Txy  = abs_vars_cur + auxvar->Txy_pos;

  float * pml_hVx  = abs_vars_rhs + auxvar->Vx_pos;
  float * pml_hVy  = abs_vars_rhs + auxvar->Vy_pos;
  float * pml_hVz  = abs_vars_rhs + auxvar->Vz_pos;
  float * pml_hTxx = abs_vars_rhs + auxvar->Txx_pos;
  float * pml_hTyy = abs_vars_rhs + auxvar->Tyy_pos;
  float * pml_hTzz = abs_vars_rhs + auxvar->Tzz_pos;
  float * pml_hTxz = abs_vars_rhs + auxvar->Txz_pos;
  float * pml_hTyz = abs_vars_rhs + auxvar->Tyz_pos;
  float * pml_hTxy = abs_vars_rhs + auxvar->Txy_pos;
  // for each dim
  if (idim == 0 ) // x direction
  {
    if(ix<abs_ni  && iy<abs_nj && iz<abs_nk)
    {
      iptr_a = iz*(abs_nj*abs_ni) + iy*abs_ni + ix;
      iptr   = (ix + abs_ni1) + (iy+abs_nj1) * siz_line + (iz+abs_nk1) * siz_slice;
      // pml coefs
      // int abs_i = ix;
      coef_D = ptr_coef_D[ix];
      coef_A = ptr_coef_A[ix];
      coef_B = ptr_coef_B[ix];
      coef_B_minus_1 = coef_B - 1.0;

      // metric
      xix = xi_x[iptr];
      xiy = xi_y[iptr];
      xiz = xi_z[iptr];

      // medium
      lam = lam3d[iptr];
      mu  =  mu3d[iptr];
      slw = slw3d[iptr];
      lam2mu = lam + 2.0 * mu;

      // xi derivatives
      M_FD_SHIFT(DxVx , Vx , iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxVy , Vy , iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxVz , Vz , iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTxx, Txx, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTyy, Tyy, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTzz, Tzz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTxz, Txz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTyz, Tyz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTxy, Txy, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);

      // combine for corr and aux vars
       hVx_rhs = slw * ( xix*DxTxx + xiy*DxTxy + xiz*DxTxz );
       hVy_rhs = slw * ( xix*DxTxy + xiy*DxTyy + xiz*DxTyz );
       hVz_rhs = slw * ( xix*DxTxz + xiy*DxTyz + xiz*DxTzz );
      hTxx_rhs = lam2mu*xix*DxVx + lam*xiy*DxVy + lam*xiz*DxVz;
      hTyy_rhs = lam*xix*DxVx + lam2mu*xiy*DxVy + lam*xiz*DxVz;
      hTzz_rhs = lam*xix*DxVx + lam*xiy*DxVy + lam2mu*xiz*DxVz;
      hTxy_rhs = mu*( xiy*DxVx + xix*DxVy );
      hTxz_rhs = mu*( xiz*DxVx + xix*DxVz );
      hTyz_rhs = mu*( xiz*DxVy + xiy*DxVz );

      // 1: make corr to moment equation
      hVx[iptr] += coef_B_minus_1 * hVx_rhs - coef_B * pml_Vx[iptr_a];
      hVy[iptr] += coef_B_minus_1 * hVy_rhs - coef_B * pml_Vy[iptr_a];
      hVz[iptr] += coef_B_minus_1 * hVz_rhs - coef_B * pml_Vz[iptr_a];

      // make corr to Hooke's equatoin
      hTxx[iptr] += coef_B_minus_1 * hTxx_rhs - coef_B * pml_Txx[iptr_a];
      hTyy[iptr] += coef_B_minus_1 * hTyy_rhs - coef_B * pml_Tyy[iptr_a];
      hTzz[iptr] += coef_B_minus_1 * hTzz_rhs - coef_B * pml_Tzz[iptr_a];
      hTxz[iptr] += coef_B_minus_1 * hTxz_rhs - coef_B * pml_Txz[iptr_a];
      hTyz[iptr] += coef_B_minus_1 * hTyz_rhs - coef_B * pml_Tyz[iptr_a];
      hTxy[iptr] += coef_B_minus_1 * hTxy_rhs - coef_B * pml_Txy[iptr_a];
      
      // 2: aux var
      //   a1 = alpha + d / beta, dealt in abs_set_cfspml
      pml_hVx[iptr_a]  = coef_D * hVx_rhs  - coef_A * pml_Vx[iptr_a];
      pml_hVy[iptr_a]  = coef_D * hVy_rhs  - coef_A * pml_Vy[iptr_a];
      pml_hVz[iptr_a]  = coef_D * hVz_rhs  - coef_A * pml_Vz[iptr_a];
      pml_hTxx[iptr_a] = coef_D * hTxx_rhs - coef_A * pml_Txx[iptr_a];
      pml_hTyy[iptr_a] = coef_D * hTyy_rhs - coef_A * pml_Tyy[iptr_a];
      pml_hTzz[iptr_a] = coef_D * hTzz_rhs - coef_A * pml_Tzz[iptr_a];
      pml_hTxz[iptr_a] = coef_D * hTxz_rhs - coef_A * pml_Txz[iptr_a];
      pml_hTyz[iptr_a] = coef_D * hTyz_rhs - coef_A * pml_Tyz[iptr_a];
      pml_hTxy[iptr_a] = coef_D * hTxy_rhs - coef_A * pml_Txy[iptr_a];

      // add contributions from free surface condition
      //  not consider timg because conflict with main cfspml,
      //     need to revise in the future if required
      if (bdryfree.is_at_sides[CONST_NDIM-1][1]==1 && (iz+abs_nk1)==nk2)
      {
        // zeta derivatives
        size_t ij = ((ix+abs_ni1) + (iy+abs_nj1) * siz_line)*9;
        Dx_DzVx = matVx2Vz[ij+3*0+0] * DxVx
                + matVx2Vz[ij+3*0+1] * DxVy
                + matVx2Vz[ij+3*0+2] * DxVz;

        Dx_DzVy = matVx2Vz[ij+3*1+0] * DxVx
                + matVx2Vz[ij+3*1+1] * DxVy
                + matVx2Vz[ij+3*1+2] * DxVz;

        Dx_DzVz = matVx2Vz[ij+3*2+0] * DxVx
                + matVx2Vz[ij+3*2+1] * DxVy
                + matVx2Vz[ij+3*2+2] * DxVz;

        // metric
        ztx = zt_x[iptr];
        zty = zt_y[iptr];
        ztz = zt_z[iptr];

        // keep xi derivative terms, including free surface convered
        hTxx_rhs =    lam2mu * (            ztx*Dx_DzVx)
                    + lam    * (            zty*Dx_DzVy
                                +           ztz*Dx_DzVz);

        hTyy_rhs =   lam2mu * (            zty*Dx_DzVy)
                    +lam    * (            ztx*Dx_DzVx
                                          +ztz*Dx_DzVz);

        hTzz_rhs =   lam2mu * (            ztz*Dx_DzVz)
                    +lam    * (            ztx*Dx_DzVx
                                          +zty*Dx_DzVy);

        hTxy_rhs = mu *(
                     zty*Dx_DzVx + ztx*Dx_DzVy
                    );
        hTxz_rhs = mu *(
                     ztz*Dx_DzVx + ztx*Dx_DzVz
                    );
        hTyz_rhs = mu *(
                     ztz*Dx_DzVy + zty*Dx_DzVz
                    );

        // make corr to Hooke's equatoin
        hTxx[iptr] += (coef_B - 1.0) * hTxx_rhs;
        hTyy[iptr] += (coef_B - 1.0) * hTyy_rhs;
        hTzz[iptr] += (coef_B - 1.0) * hTzz_rhs;
        hTxz[iptr] += (coef_B - 1.0) * hTxz_rhs;
        hTyz[iptr] += (coef_B - 1.0) * hTyz_rhs;
        hTxy[iptr] += (coef_B - 1.0) * hTxy_rhs;

        // aux var
        //   a1 = alpha + d / beta, dealt in abs_set_cfspml
        pml_hTxx[iptr_a] += coef_D * hTxx_rhs;
        pml_hTyy[iptr_a] += coef_D * hTyy_rhs;
        pml_hTzz[iptr_a] += coef_D * hTzz_rhs;
        pml_hTxz[iptr_a] += coef_D * hTxz_rhs;
        pml_hTyz[iptr_a] += coef_D * hTyz_rhs;
        pml_hTxy[iptr_a] += coef_D * hTxy_rhs;
      }
    }
  }
  else if (idim == 1) // y direction
  {
    if(ix<abs_ni  && iy<abs_nj && iz<abs_nk)
    {
      iptr_a = iz*(abs_nj*abs_ni) + iy*abs_ni + ix;
      iptr   = (ix + abs_ni1) + (iy+abs_nj1)*siz_line + (iz+abs_nk1) * siz_slice;

      // pml coefs
      // int abs_j = iy;
      coef_D = ptr_coef_D[iy];
      coef_A = ptr_coef_A[iy];
      coef_B = ptr_coef_B[iy];
      coef_B_minus_1 = coef_B - 1.0;

      // metric
      etx = et_x[iptr];
      ety = et_y[iptr];
      etz = et_z[iptr];

      // medium
      lam = lam3d[iptr];
      mu  =  mu3d[iptr];
      slw = slw3d[iptr];
      lam2mu = lam + 2.0 * mu;

      // et derivatives
      M_FD_SHIFT(DyVx , Vx , iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyVy , Vy , iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyVz , Vz , iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTxx, Txx, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTyy, Tyy, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTzz, Tzz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTxz, Txz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTyz, Tyz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTxy, Txy, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      // combine for corr and aux vars
       hVx_rhs = slw * ( etx*DyTxx + ety*DyTxy + etz*DyTxz );
       hVy_rhs = slw * ( etx*DyTxy + ety*DyTyy + etz*DyTyz );
       hVz_rhs = slw * ( etx*DyTxz + ety*DyTyz + etz*DyTzz );
      hTxx_rhs = lam2mu*etx*DyVx + lam*ety*DyVy + lam*etz*DyVz;
      hTyy_rhs = lam*etx*DyVx + lam2mu*ety*DyVy + lam*etz*DyVz;
      hTzz_rhs = lam*etx*DyVx + lam*ety*DyVy + lam2mu*etz*DyVz;
      hTxy_rhs = mu*( ety*DyVx + etx*DyVy );
      hTxz_rhs = mu*( etz*DyVx + etx*DyVz );
      hTyz_rhs = mu*( etz*DyVy + ety*DyVz );

      // 1: make corr to moment equation
      hVx[iptr] += coef_B_minus_1 * hVx_rhs - coef_B * pml_Vx[iptr_a];
      hVy[iptr] += coef_B_minus_1 * hVy_rhs - coef_B * pml_Vy[iptr_a];
      hVz[iptr] += coef_B_minus_1 * hVz_rhs - coef_B * pml_Vz[iptr_a];

      // make corr to Hooke's equatoin
      hTxx[iptr] += coef_B_minus_1 * hTxx_rhs - coef_B * pml_Txx[iptr_a];
      hTyy[iptr] += coef_B_minus_1 * hTyy_rhs - coef_B * pml_Tyy[iptr_a];
      hTzz[iptr] += coef_B_minus_1 * hTzz_rhs - coef_B * pml_Tzz[iptr_a];
      hTxz[iptr] += coef_B_minus_1 * hTxz_rhs - coef_B * pml_Txz[iptr_a];
      hTyz[iptr] += coef_B_minus_1 * hTyz_rhs - coef_B * pml_Tyz[iptr_a];
      hTxy[iptr] += coef_B_minus_1 * hTxy_rhs - coef_B * pml_Txy[iptr_a];
      
      // 2: aux var
      //   a1 = alpha + d / beta, dealt in abs_set_cfspml
      pml_hVx[iptr_a]  = coef_D * hVx_rhs  - coef_A * pml_Vx[iptr_a];
      pml_hVy[iptr_a]  = coef_D * hVy_rhs  - coef_A * pml_Vy[iptr_a];
      pml_hVz[iptr_a]  = coef_D * hVz_rhs  - coef_A * pml_Vz[iptr_a];
      pml_hTxx[iptr_a] = coef_D * hTxx_rhs - coef_A * pml_Txx[iptr_a];
      pml_hTyy[iptr_a] = coef_D * hTyy_rhs - coef_A * pml_Tyy[iptr_a];
      pml_hTzz[iptr_a] = coef_D * hTzz_rhs - coef_A * pml_Tzz[iptr_a];
      pml_hTxz[iptr_a] = coef_D * hTxz_rhs - coef_A * pml_Txz[iptr_a];
      pml_hTyz[iptr_a] = coef_D * hTyz_rhs - coef_A * pml_Tyz[iptr_a];
      pml_hTxy[iptr_a] = coef_D * hTxy_rhs - coef_A * pml_Txy[iptr_a];

      // add contributions from free surface condition
      if (bdryfree.is_at_sides[CONST_NDIM-1][1]==1 && (iz+abs_nk1)==nk2)
      {
        // zeta derivatives
        size_t ij = ((ix+abs_ni1) + (iy+abs_nj1) * siz_line)*9;
        Dy_DzVx = matVy2Vz[ij+3*0+0] * DyVx
                + matVy2Vz[ij+3*0+1] * DyVy
                + matVy2Vz[ij+3*0+2] * DyVz;

        Dy_DzVy = matVy2Vz[ij+3*1+0] * DyVx
                + matVy2Vz[ij+3*1+1] * DyVy
                + matVy2Vz[ij+3*1+2] * DyVz;

        Dy_DzVz = matVy2Vz[ij+3*2+0] * DyVx
                + matVy2Vz[ij+3*2+1] * DyVy
                + matVy2Vz[ij+3*2+2] * DyVz;

        // metric
        ztx = zt_x[iptr];
        zty = zt_y[iptr];
        ztz = zt_z[iptr];

        hTxx_rhs =    lam2mu * (             ztx*Dy_DzVx)
                    + lam    * (             zty*Dy_DzVy
                                            +ztz*Dy_DzVz);

        hTyy_rhs =   lam2mu * (             zty*Dy_DzVy)
                    +lam    * (             ztx*Dy_DzVx
                                           +ztz*Dy_DzVz);

        hTzz_rhs =   lam2mu * (             ztz*Dy_DzVz)
                    +lam    * (             ztx*Dy_DzVx
                                           +zty*Dy_DzVy);

        hTxy_rhs = mu *(
                     zty*Dy_DzVx + ztx*Dy_DzVy
                    );
        hTxz_rhs = mu *(
                     ztz*Dy_DzVx + ztx*Dy_DzVz
                    );
        hTyz_rhs = mu *(
                     ztz*Dy_DzVy + zty*Dy_DzVz
                  );

        // make corr to Hooke's equatoin
        hTxx[iptr] += (coef_B - 1.0) * hTxx_rhs;
        hTyy[iptr] += (coef_B - 1.0) * hTyy_rhs;
        hTzz[iptr] += (coef_B - 1.0) * hTzz_rhs;
        hTxz[iptr] += (coef_B - 1.0) * hTxz_rhs;
        hTyz[iptr] += (coef_B - 1.0) * hTyz_rhs;
        hTxy[iptr] += (coef_B - 1.0) * hTxy_rhs;

        // aux var
        //   a1 = alpha + d / beta, dealt in abs_set_cfspml
        pml_hTxx[iptr_a] += coef_D * hTxx_rhs;
        pml_hTyy[iptr_a] += coef_D * hTyy_rhs;
        pml_hTzz[iptr_a] += coef_D * hTzz_rhs;
        pml_hTxz[iptr_a] += coef_D * hTxz_rhs;
        pml_hTyz[iptr_a] += coef_D * hTyz_rhs;
        pml_hTxy[iptr_a] += coef_D * hTxy_rhs;
      }
    }
  }
  else // z direction
  {
    if(ix<abs_ni  && iy<abs_nj && iz<abs_nk)
    {
      iptr_a = iz*(abs_nj*abs_ni) + iy*abs_ni + ix;
      iptr   = (ix + abs_ni1) + (iy+abs_nj1) * siz_line + (iz+abs_nk1) * siz_slice;
      // pml coefs
      // int abs_k = iz;
      coef_D = ptr_coef_D[iz];
      coef_A = ptr_coef_A[iz];
      coef_B = ptr_coef_B[iz];
      coef_B_minus_1 = coef_B - 1.0;

      // metric
      ztx = zt_x[iptr];
      zty = zt_y[iptr];
      ztz = zt_z[iptr];

      // medium
      lam = lam3d[iptr];
      mu  =  mu3d[iptr];
      slw = slw3d[iptr];
      lam2mu = lam + 2.0 * mu;

      // zt derivatives
      M_FD_SHIFT(DzVx , Vx , iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzVy , Vy , iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzVz , Vz , iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTxx, Txx, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTyy, Tyy, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTzz, Tzz, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTxz, Txz, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTyz, Tyz, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTxy, Txy, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);

      // combine for corr and aux vars
       hVx_rhs = slw * ( ztx*DzTxx + zty*DzTxy + ztz*DzTxz );
       hVy_rhs = slw * ( ztx*DzTxy + zty*DzTyy + ztz*DzTyz );
       hVz_rhs = slw * ( ztx*DzTxz + zty*DzTyz + ztz*DzTzz );
      hTxx_rhs = lam2mu*ztx*DzVx + lam*zty*DzVy + lam*ztz*DzVz;
      hTyy_rhs = lam*ztx*DzVx + lam2mu*zty*DzVy + lam*ztz*DzVz;
      hTzz_rhs = lam*ztx*DzVx + lam*zty*DzVy + lam2mu*ztz*DzVz;
      hTxy_rhs = mu*( zty*DzVx + ztx*DzVy );
      hTxz_rhs = mu*( ztz*DzVx + ztx*DzVz );
      hTyz_rhs = mu*( ztz*DzVy + zty*DzVz );

      // 1: make corr to moment equation
      hVx[iptr] += coef_B_minus_1 * hVx_rhs - coef_B * pml_Vx[iptr_a];
      hVy[iptr] += coef_B_minus_1 * hVy_rhs - coef_B * pml_Vy[iptr_a];
      hVz[iptr] += coef_B_minus_1 * hVz_rhs - coef_B * pml_Vz[iptr_a];

      // make corr to Hooke's equatoin
      hTxx[iptr] += coef_B_minus_1 * hTxx_rhs - coef_B * pml_Txx[iptr_a];
      hTyy[iptr] += coef_B_minus_1 * hTyy_rhs - coef_B * pml_Tyy[iptr_a];
      hTzz[iptr] += coef_B_minus_1 * hTzz_rhs - coef_B * pml_Tzz[iptr_a];
      hTxz[iptr] += coef_B_minus_1 * hTxz_rhs - coef_B * pml_Txz[iptr_a];
      hTyz[iptr] += coef_B_minus_1 * hTyz_rhs - coef_B * pml_Tyz[iptr_a];
      hTxy[iptr] += coef_B_minus_1 * hTxy_rhs - coef_B * pml_Txy[iptr_a];
      
      // 2: aux var
      //   a1 = alpha + d / beta, dealt in abs_set_cfspml
      pml_hVx[iptr_a]  = coef_D * hVx_rhs  - coef_A * pml_Vx[iptr_a];
      pml_hVy[iptr_a]  = coef_D * hVy_rhs  - coef_A * pml_Vy[iptr_a];
      pml_hVz[iptr_a]  = coef_D * hVz_rhs  - coef_A * pml_Vz[iptr_a];
      pml_hTxx[iptr_a] = coef_D * hTxx_rhs - coef_A * pml_Txx[iptr_a];
      pml_hTyy[iptr_a] = coef_D * hTyy_rhs - coef_A * pml_Tyy[iptr_a];
      pml_hTzz[iptr_a] = coef_D * hTzz_rhs - coef_A * pml_Tzz[iptr_a];
      pml_hTxz[iptr_a] = coef_D * hTxz_rhs - coef_A * pml_Txz[iptr_a];
      pml_hTyz[iptr_a] = coef_D * hTyz_rhs - coef_A * pml_Tyz[iptr_a];
      pml_hTxy[iptr_a] = coef_D * hTxy_rhs - coef_A * pml_Txy[iptr_a];
    }
  } // if which dim

  return;
}
/*******************************************************************************
 * add source terms
 ******************************************************************************/

__global__ void
sv_eq1st_curv_col_el_iso_rhs_src_gpu(
    float * hVx , float * hVy , float * hVz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * jac3d, float * slw3d,
    src_t src, // short nation for reference member
    const int myid, const int verbose)
{
  // for easy coding and efficiency
  int max_ext = src.max_ext;

  // get fi / mij
  float fx, fy, fz;
  float Mxx,Myy,Mzz,Mxz,Myz,Mxy;

  int it     = src.it;
  int istage = src.istage;
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;

  // add src; ix is a commont iterater var
  if(ix<src.total_number)
  {
    int   it_start = src.it_begin[ix];
    int   it_end   = src.it_end  [ix];

    if (it >= it_start && it <= it_end)
    {
      int   *ptr_ext_indx = src.ext_indx + ix * max_ext;
      float *ptr_ext_coef = src.ext_coef + ix * max_ext;
      int it_to_it_start = it - it_start;
      size_t iptr_cur_stage =   ix * src.max_nt * src.max_stage // skip other src
                           + it_to_it_start * src.max_stage // skip other time step
                           + istage;
      if (src.force_actived == 1) {
        fx  = src.Fx [iptr_cur_stage];
        fy  = src.Fy [iptr_cur_stage];
        fz  = src.Fz [iptr_cur_stage];
      }
      if (src.moment_actived == 1) {
        Mxx = src.Mxx[iptr_cur_stage];
        Myy = src.Myy[iptr_cur_stage];
        Mzz = src.Mzz[iptr_cur_stage];
        Mxz = src.Mxz[iptr_cur_stage];
        Myz = src.Myz[iptr_cur_stage];
        Mxy = src.Mxy[iptr_cur_stage];
      }
      
      // for extend points
      for (int i_ext=0; i_ext < src.ext_num[ix]; i_ext++)
      {
        int   iptr = ptr_ext_indx[i_ext];
        float coef = ptr_ext_coef[i_ext];
        if (src.force_actived == 1) {
          float V = coef * slw3d[iptr] / jac3d[iptr];
          atomicAdd(&hVx[iptr], fx * V);
          atomicAdd(&hVy[iptr], fy * V);
          atomicAdd(&hVz[iptr], fz * V);
        }

        if (src.moment_actived == 1) {
          float rjac = coef / jac3d[iptr];
          atomicAdd(&hTxx[iptr], -Mxx * rjac);
          atomicAdd(&hTyy[iptr], -Myy * rjac);
          atomicAdd(&hTzz[iptr], -Mzz * rjac);
          atomicAdd(&hTxz[iptr], -Mxz * rjac);
          atomicAdd(&hTyz[iptr], -Myz * rjac);
          atomicAdd(&hTxy[iptr], -Mxy * rjac);
        }
      } // i_ext
    } // it
  }

  return;
}

/*******************************************************************************
 * free surface coef
 ******************************************************************************/

__global__ void
sv_eq1st_curv_col_el_iso_dvh2dvz_gpu(gdinfo_t        gdinfo_d,
                                     gdcurv_metric_t metric_d,
                                     md_t       md_d,
                                     bdryfree_t      bdryfree_d,
                                     const int verbose)
{
  int ni1 = gdinfo_d.ni1;
  int ni2 = gdinfo_d.ni2;
  int nj1 = gdinfo_d.nj1;
  int nj2 = gdinfo_d.nj2;
  int nk1 = gdinfo_d.nk1;
  int nk2 = gdinfo_d.nk2;
  int nx  = gdinfo_d.nx;
  int ny  = gdinfo_d.ny;
  int nz  = gdinfo_d.nz;
  size_t siz_line   = gdinfo_d.siz_iy;
  size_t siz_slice  = gdinfo_d.siz_iz;
  size_t siz_volume = gdinfo_d.siz_icmp;

  // point to each var
  float * xi_x = metric_d.xi_x;
  float * xi_y = metric_d.xi_y;
  float * xi_z = metric_d.xi_z;
  float * et_x = metric_d.eta_x;
  float * et_y = metric_d.eta_y;
  float * et_z = metric_d.eta_z;
  float * zt_x = metric_d.zeta_x;
  float * zt_y = metric_d.zeta_y;
  float * zt_z = metric_d.zeta_z;

  float * lam3d = md_d.lambda;
  float *  mu3d = md_d.mu;

  float *matVx2Vz = bdryfree_d.matVx2Vz2;
  float *matVy2Vz = bdryfree_d.matVy2Vz2;
  
  float A[3][3], B[3][3], C[3][3];
  float AB[3][3], AC[3][3];

  float e11, e12, e13, e21, e22, e23, e31, e32, e33;
  float lam2mu, lam, mu;
 
  int k = nk2;
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  if(ix<(ni2-ni1+1) && iy<(nj2-nj1+1))
  {
    size_t iptr = (ix+ni1) + (iy+nj1) * siz_line + k * siz_slice;
    e11 = xi_x[iptr];
    e12 = xi_y[iptr];
    e13 = xi_z[iptr];
    e21 = et_x[iptr];
    e22 = et_y[iptr];
    e23 = et_z[iptr];
    e31 = zt_x[iptr];
    e32 = zt_y[iptr];
    e33 = zt_z[iptr];

    lam    = lam3d[iptr];
    mu     =  mu3d[iptr];
    lam2mu = lam + 2.0f * mu;

    // first dim: irow; sec dim: jcol, as Fortran code
    A[0][0] = lam2mu*e31*e31 + mu*(e32*e32+e33*e33);
    A[0][1] = lam*e31*e32 + mu*e32*e31;
    A[0][2] = lam*e31*e33 + mu*e33*e31;
    A[1][0] = lam*e32*e31 + mu*e31*e32;
    A[1][1] = lam2mu*e32*e32 + mu*(e31*e31+e33*e33);
    A[1][2] = lam*e32*e33 + mu*e33*e32;
    A[2][0] = lam*e33*e31 + mu*e31*e33;
    A[2][1] = lam*e33*e32 + mu*e32*e33;
    A[2][2] = lam2mu*e33*e33 + mu*(e31*e31+e32*e32);
    fdlib_math_invert3x3(A);

    B[0][0] = -lam2mu*e31*e11 - mu*(e32*e12+e33*e13);
    B[0][1] = -lam*e31*e12 - mu*e32*e11;
    B[0][2] = -lam*e31*e13 - mu*e33*e11;
    B[1][0] = -lam*e32*e11 - mu*e31*e12;
    B[1][1] = -lam2mu*e32*e12 - mu*(e31*e11+e33*e13);
    B[1][2] = -lam*e32*e13 - mu*e33*e12;
    B[2][0] = -lam*e33*e11 - mu*e31*e13;
    B[2][1] = -lam*e33*e12 - mu*e32*e13;
    B[2][2] = -lam2mu*e33*e13 - mu*(e31*e11+e32*e12);

    C[0][0] = -lam2mu*e31*e21 - mu*(e32*e22+e33*e23);
    C[0][1] = -lam*e31*e22 - mu*e32*e21;
    C[0][2] = -lam*e31*e23 - mu*e33*e21;
    C[1][0] = -lam*e32*e21 - mu*e31*e22;
    C[1][1] = -lam2mu*e32*e22 - mu*(e31*e21+e33*e23);
    C[1][2] = -lam*e32*e23 - mu*e33*e22;
    C[2][0] = -lam*e33*e21 - mu*e31*e23;
    C[2][1] = -lam*e33*e22 - mu*e32*e23;
    C[2][2] = -lam2mu*e33*e23 - mu*(e31*e21+e32*e22);

    fdlib_math_matmul3x3(A, B, AB);
    fdlib_math_matmul3x3(A, C, AC);

    size_t ij = ((iy+nj1) * siz_line + (ix+ni1)) * 9;

    // save into mat
    for(int irow = 0; irow < 3; irow++){
      for(int jcol = 0; jcol < 3; jcol++){
        matVx2Vz[ij + irow*3 + jcol] = AB[irow][jcol];
        matVy2Vz[ij + irow*3 + jcol] = AC[irow][jcol];
      }
    }
  }

  return;
}
