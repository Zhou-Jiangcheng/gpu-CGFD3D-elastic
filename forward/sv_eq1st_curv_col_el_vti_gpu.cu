#include "hip/hip_runtime.h"
/*******************************************************************************
 * solver of isotropic elastic 1st-order eqn using curv grid and macdrp schem
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <mpi.h>

#include "fdlib_mem.h"
#include "fdlib_math.h"
#include "sv_eq1st_curv_col_el_iso_gpu.h"
#include "sv_eq1st_curv_col_el_vti_gpu.h"
#include "cuda_common.h"

/*******************************************************************************
 * perform one stage calculation of rhs
 ******************************************************************************/

void
sv_eq1st_curv_col_el_vti_onestage(
  float *w_cur_d,
  float *rhs_d, 
  wav_t  wav_d,
  fd_wav_t fd_wav_d,
  gdinfo_t  gdinfo_d,
  gdcurv_metric_t metric_d,
  md_t md_d,
  bdryfree_t bdryfree_d,
  bdrypml_t  bdrypml_d,
  src_t src_d,
  // include different order/stentil
  int num_of_fdx_op, fd_op_t *fdx_op,
  int num_of_fdy_op, fd_op_t *fdy_op,
  int num_of_fdz_op, fd_op_t *fdz_op,
  int fdz_max_len, 
  const int myid, const int verbose)
{
  // local pointer get each vars
  float *Vx    = w_cur_d + wav_d.Vx_pos ;
  float *Vy    = w_cur_d + wav_d.Vy_pos ;
  float *Vz    = w_cur_d + wav_d.Vz_pos ;
  float *Txx   = w_cur_d + wav_d.Txx_pos;
  float *Tyy   = w_cur_d + wav_d.Tyy_pos;
  float *Tzz   = w_cur_d + wav_d.Tzz_pos;
  float *Txz   = w_cur_d + wav_d.Txz_pos;
  float *Tyz   = w_cur_d + wav_d.Tyz_pos;
  float *Txy   = w_cur_d + wav_d.Txy_pos;
  float *hVx   = rhs_d   + wav_d.Vx_pos ; 
  float *hVy   = rhs_d   + wav_d.Vy_pos ; 
  float *hVz   = rhs_d   + wav_d.Vz_pos ; 
  float *hTxx  = rhs_d   + wav_d.Txx_pos; 
  float *hTyy  = rhs_d   + wav_d.Tyy_pos; 
  float *hTzz  = rhs_d   + wav_d.Tzz_pos; 
  float *hTxz  = rhs_d   + wav_d.Txz_pos; 
  float *hTyz  = rhs_d   + wav_d.Tyz_pos; 
  float *hTxy  = rhs_d   + wav_d.Txy_pos; 

  float *xi_x  = metric_d.xi_x;
  float *xi_y  = metric_d.xi_y;
  float *xi_z  = metric_d.xi_z;
  float *et_x  = metric_d.eta_x;
  float *et_y  = metric_d.eta_y;
  float *et_z  = metric_d.eta_z;
  float *zt_x  = metric_d.zeta_x;
  float *zt_y  = metric_d.zeta_y;
  float *zt_z  = metric_d.zeta_z;
  float *jac3d = metric_d.jac;

  float *c11   = md_d.c11;
  float *c13   = md_d.c13;
  float *c33   = md_d.c33;
  float *c55   = md_d.c55;
  float *c66   = md_d.c66;
  float *slw3d = md_d.rho;

  // grid size
  int ni1 = gdinfo_d.ni1;
  int ni2 = gdinfo_d.ni2;
  int nj1 = gdinfo_d.nj1;
  int nj2 = gdinfo_d.nj2;
  int nk1 = gdinfo_d.nk1;
  int nk2 = gdinfo_d.nk2;

  int ni  = gdinfo_d.ni;
  int nj  = gdinfo_d.nj;
  int nk  = gdinfo_d.nk;
  int nx  = gdinfo_d.nx;
  int ny  = gdinfo_d.ny;
  int nz  = gdinfo_d.nz;
  size_t siz_line   = gdinfo_d.siz_line;
  size_t siz_slice  = gdinfo_d.siz_slice;
  size_t siz_volume = gdinfo_d.siz_volume;

  float *matVx2Vz = bdryfree_d.matVx2Vz2;
  float *matVy2Vz = bdryfree_d.matVy2Vz2;

  // local fd op
  int    fdx_len;
  int    *fdx_indx;
  float  *fdx_coef;
  int    fdy_len;
  int    *fdy_indx;
  float  *fdy_coef;
  int    fdz_len;
  int    *fdz_indx;
  float  *fdz_coef;

  // for get a op from 1d array, currently use num_of_fdz_op as index
  // length, index, coef of a op
  fdx_len  = fdx_op[num_of_fdx_op-1].total_len;
  fdx_indx = fdx_op[num_of_fdx_op-1].indx;
  fdx_coef = fdx_op[num_of_fdx_op-1].coef;

  fdy_len  = fdy_op[num_of_fdy_op-1].total_len;
  fdy_indx = fdy_op[num_of_fdy_op-1].indx;
  fdy_coef = fdy_op[num_of_fdy_op-1].coef;

  fdz_len  = fdz_op[num_of_fdz_op-1].total_len;
  fdz_indx = fdz_op[num_of_fdz_op-1].indx;
  fdz_coef = fdz_op[num_of_fdz_op-1].coef;

  // use local stack array for speedup
  float  lfdx_coef [fdx_len];
  size_t lfdx_shift[fdx_len];
  float  lfdy_coef [fdy_len];
  size_t lfdy_shift[fdy_len];
  float  lfdz_coef [fdz_len];
  size_t lfdz_shift[fdz_len];

  // put fd op into local array
  for (int i=0; i < fdx_len; i++) {
    lfdx_coef [i] = fdx_coef[i];
    lfdx_shift[i] = fdx_indx[i];
  }
  for (int j=0; j < fdy_len; j++) {
    lfdy_coef [j] = fdy_coef[j];
    lfdy_shift[j] = fdy_indx[j] * siz_line;
  }
  for (int k=0; k < fdz_len; k++) {
    lfdz_coef [k] = fdz_coef[k];
    lfdz_shift[k] = fdz_indx[k] * siz_slice;
  }

  // allocate max_len because fdz may have different lens
  // these array is for low order surface
  float  fdz_coef_all [num_of_fdz_op*fdz_max_len];
  size_t fdz_shift_all[num_of_fdz_op*fdz_max_len];
  int    fdz_len_all[num_of_fdz_op];
  // loop near surface layers
  for (int n=0; n < num_of_fdz_op-1; n++)
  {
    // get pos and len for this point
    fdz_len_all[n]  = fdz_op[n].total_len;
    // point to indx/coef for this point
    int   *p_fdz_indx  = fdz_op[n].indx;
    float *p_fdz_coef  = fdz_op[n].coef;
    for (int n_fd = 0; n_fd < fdz_len_all[n] ; n_fd++) {
      fdz_shift_all[n_fd + n*fdz_max_len]  = p_fdz_indx[n_fd] * siz_slice;
      fdz_coef_all [n_fd + n*fdz_max_len]  = p_fdz_coef[n_fd];
    }
  }

  int  *lfdz_len_d = fd_wav_d.fdz_len_d;
  float *lfdx_coef_d = fd_wav_d.fdx_coef_d;
  float *lfdy_coef_d = fd_wav_d.fdy_coef_d;
  float *lfdz_coef_d = fd_wav_d.fdz_coef_d;
  float *lfdz_coef_all_d = fd_wav_d.fdz_coef_all_d;
  size_t  *lfdx_shift_d = fd_wav_d.fdx_shift_d;
  size_t  *lfdy_shift_d = fd_wav_d.fdy_shift_d;
  size_t  *lfdz_shift_d = fd_wav_d.fdz_shift_d;
  size_t  *lfdz_shift_all_d = fd_wav_d.fdz_shift_all_d;
  int  *lfdx_indx_d = fd_wav_d.fdx_indx_d;
  int  *lfdy_indx_d = fd_wav_d.fdy_indx_d;
  int  *lfdz_indx_d = fd_wav_d.fdz_indx_d;
  int  *lfdz_indx_all_d = fd_wav_d.fdz_indx_all_d;
  //host to device
  CUDACHECK(hipMemcpy(lfdx_coef_d,lfdx_coef,fdx_len*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdy_coef_d,lfdy_coef,fdy_len*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_coef_d,lfdz_coef,fdz_len*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdx_shift_d,lfdx_shift,fdx_len*sizeof(size_t),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdy_shift_d,lfdy_shift,fdy_len*sizeof(size_t),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_shift_d,lfdz_shift,fdz_len*sizeof(size_t),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdx_indx_d,fdx_indx,fdx_len*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdy_indx_d,fdy_indx,fdy_len*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_indx_d,fdz_indx,fdz_len*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_len_d,fdz_len_all,num_of_fdz_op*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_coef_all_d,fdz_coef_all,fdz_max_len*num_of_fdz_op*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_shift_all_d,fdz_shift_all,fdz_max_len*num_of_fdz_op*sizeof(size_t),hipMemcpyHostToDevice));

  {
    dim3 block(8,8,8);
    dim3 grid;
    grid.x = (ni+block.x-1)/block.x;
    grid.y = (nj+block.y-1)/block.y;
    grid.z = (nk+block.z-1)/block.z;
    sv_eq1st_curv_col_el_vti_rhs_inner_gpu <<<grid, block>>> (
                        Vx,Vy,Vz,Txx,Tyy,Tzz,Txz,Tyz,Txy,
                        hVx,hVy,hVz,hTxx,hTyy,hTzz,hTxz,hTyz,hTxy,
                        xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                        c11, c13, c33, c55, c66, slw3d,
                        ni1,ni,nj1,nj,nk1,nk,siz_line,siz_slice,
                        fdx_len, lfdx_shift_d, lfdx_coef_d,
                        fdy_len, lfdy_shift_d, lfdy_coef_d,
                        fdz_len, lfdz_shift_d, lfdz_coef_d,
                        myid, verbose);
  }

  // free surface at z2
  if (bdryfree_d.is_at_sides[2][1] == 1)
  {
    // tractiong
    {
      dim3 block(8,8);
      dim3 grid;
      grid.x = (ni+block.x-1)/block.x;
      grid.y = (nj+block.y-1)/block.y;
      sv_eq1st_curv_col_el_iso_rhs_timg_z2_gpu  <<<grid, block>>> (
                          Txx,Tyy,Tzz,Txz,Tyz,Txy,hVx,hVy,hVz,
                          xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                          jac3d, slw3d,
                          ni1,ni,nj1,nj,nk1,nk2,siz_line,siz_slice,
                          fdx_len, lfdx_indx_d, lfdx_coef_d,
                          fdy_len, lfdy_indx_d, lfdy_coef_d,
                          fdz_len, lfdz_indx_d, lfdz_coef_d,
                          myid, verbose);
      hipDeviceSynchronize();
    }
    // velocity: vlow
    {
      dim3 block(8,8);
      dim3 grid;
      grid.x = (ni+block.x-1)/block.x;
      grid.y = (nj+block.y-1)/block.y;
      sv_eq1st_curv_col_el_vti_rhs_vlow_z2_gpu  <<<grid, block>>> (
                        Vx,Vy,Vz,hTxx,hTyy,hTzz,hTxz,hTyz,hTxy,
                        xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                        c11, c13, c33, c55, c66, slw3d,
                        matVx2Vz,matVy2Vz,
                        ni1,ni,nj1,nj,nk1,nk2,siz_line,siz_slice,
                        fdx_len, lfdx_shift_d, lfdx_coef_d,
                        fdy_len, lfdy_shift_d, lfdy_coef_d,
                        num_of_fdz_op,fdz_max_len,lfdz_len_d,
                        lfdz_coef_all_d,lfdz_shift_all_d,
                        myid, verbose);
      CUDACHECK( hipDeviceSynchronize() );
    }
  }

  // cfs-pml, loop face inside
  if (bdrypml_d.is_enable == 1)
  {
    sv_eq1st_curv_col_el_vti_rhs_cfspml(Vx,Vy,Vz,Txx,Tyy,Tzz,Txz,Tyz,Txy,
                                        hVx,hVy,hVz,hTxx,hTyy,hTzz,hTxz,hTyz,hTxy,
                                        xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                                        c11, c13, c33, c55, c66, slw3d,
                                        nk2, siz_line,siz_slice,
                                        fdx_len, lfdx_shift_d, lfdx_coef_d,
                                        fdy_len, lfdy_shift_d, lfdy_coef_d,
                                        fdz_len, lfdz_shift_d, lfdz_coef_d,
                                        bdrypml_d, bdryfree_d,
                                        myid, verbose);
  }

  // add source term
  if (src_d.total_number > 0)
  {
    {
      dim3 block(256);
      dim3 grid;
      grid.x = (src_d.total_number+block.x-1) / block.x;
      sv_eq1st_curv_col_el_iso_rhs_src_gpu  <<< grid,block >>> (
                        hVx, hVy, hVz, hTxx, hTyy, hTzz, hTxz, hTyz, hTxy,
                        jac3d, slw3d, 
                        src_d,
                        myid, verbose);
      CUDACHECK( hipDeviceSynchronize() );
    }
  }

  return;
}

/*******************************************************************************
 * calculate all points without boundaries treatment
 ******************************************************************************/

__global__ void
sv_eq1st_curv_col_el_vti_rhs_inner_gpu(
    float *  Vx , float *  Vy , float *  Vz ,
    float *  Txx, float *  Tyy, float *  Tzz,
    float *  Txz, float *  Tyz, float *  Txy,
    float * hVx , float * hVy , float * hVz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float *c11d, float *c13d, float *c33d,
    float *c55d, float *c66d, float *slw3d,
    int ni1, int ni, int nj1, int nj, int nk1, int nk,
    size_t siz_line, size_t siz_slice,
    int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
    int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
    int fdz_len, size_t * lfdz_shift, float * lfdz_coef,
    const int myid, const int verbose)
{
  // local var
  float DxTxx,DxTyy,DxTzz,DxTxy,DxTxz,DxTyz,DxVx,DxVy,DxVz;
  float DyTxx,DyTyy,DyTzz,DyTxy,DyTxz,DyTyz,DyVx,DyVy,DyVz;
  float DzTxx,DzTyy,DzTzz,DzTxy,DzTxz,DzTyz,DzVx,DzVy,DzVz;
  float c11,c13,c33,c55,c66,c12,slw;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;

  float * Vx_ptr;
  float * Vy_ptr;
  float * Vz_ptr;
  float * Txx_ptr;
  float * Txy_ptr;
  float * Txz_ptr;
  float * Tyy_ptr;
  float * Tzz_ptr;
  float * Tyz_ptr;


  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  size_t iz = blockIdx.z * blockDim.z + threadIdx.z;

  // caclu all points
  if(ix<ni && iy<nj && iz<nk)
  {
    size_t iptr = (ix+ni1) + (iy+nj1) * siz_line + (iz+nk1) * siz_slice;

    Vx_ptr = Vx + iptr;
    Vy_ptr = Vy + iptr;
    Vz_ptr = Vz + iptr;
    Txx_ptr = Txx + iptr;
    Tyy_ptr = Tyy + iptr;
    Tzz_ptr = Tzz + iptr;
    Txz_ptr = Txz + iptr;
    Tyz_ptr = Tyz + iptr;
    Txy_ptr = Txy + iptr;

    // Vx derivatives
    M_FD_SHIFT_PTR_MACDRP(DxVx, Vx_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyVx, Vx_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzVx, Vx_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Vy derivatives
    M_FD_SHIFT_PTR_MACDRP(DxVy, Vy_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyVy, Vy_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzVy, Vy_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Vz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxVz, Vz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyVz, Vz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzVz, Vz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Txx derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTxx, Txx_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTxx, Txx_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTxx, Txx_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Tyy derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTyy, Tyy_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTyy, Tyy_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTyy, Tyy_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Tzz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTzz, Tzz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTzz, Tzz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTzz, Tzz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Txz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTxz, Txz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTxz, Txz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTxz, Txz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Tyz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTyz, Tyz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTyz, Tyz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTyz, Tyz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Txy derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTxy, Txy_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTxy, Txy_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTxy, Txy_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // metric
    xix = xi_x[iptr];
    xiy = xi_y[iptr];
    xiz = xi_z[iptr];
    etx = et_x[iptr];
    ety = et_y[iptr];
    etz = et_z[iptr];
    ztx = zt_x[iptr];
    zty = zt_y[iptr];
    ztz = zt_z[iptr];

    // medium
    slw = slw3d[iptr];
    c11 = c11d[iptr];
    c13 = c13d[iptr];
    c33 = c33d[iptr];
    c55 = c55d[iptr];
    c66 = c66d[iptr];
    c12 = c11 - 2.0 * c66;

    // moment equation
    hVx[iptr] = slw*( xix*DxTxx + xiy*DxTxy + xiz*DxTxz  
                     +etx*DyTxx + ety*DyTxy + etz*DyTxz 
                     +ztx*DzTxx + zty*DzTxy + ztz*DzTxz );
    hVy[iptr] = slw*( xix*DxTxy + xiy*DxTyy + xiz*DxTyz
                     +etx*DyTxy + ety*DyTyy + etz*DyTyz
                     +ztx*DzTxy + zty*DzTyy + ztz*DzTyz );
    hVz[iptr] = slw*( xix*DxTxz + xiy*DxTyz + xiz*DxTzz 
                     +etx*DyTxz + ety*DyTyz + etz*DyTzz
                     +ztx*DzTxz + zty*DzTyz + ztz*DzTzz );

    // Hooke's equatoin

	  hTxx[iptr] = (c11*xix ) * DxVx + ( c12*xiy ) * DxVy  + ( c13*xiz) * DxVz
               + (c11*etx ) * DyVx + ( c12*ety ) * DyVy  + ( c13*etz) * DyVz
               + (c11*ztx ) * DzVx + ( c12*zty ) * DzVy  + ( c13*ztz) * DzVz;
    
    hTyy[iptr] = (c12*xix ) * DxVx + ( c11*xiy ) * DxVy + ( c13*xiz) * DxVz
               + (c12*etx ) * DyVx + ( c11*ety ) * DyVy + ( c13*etz) * DyVz
               + (c12*ztx ) * DzVx + ( c11*zty ) * DzVy + ( c13*ztz) * DzVz;
    
    hTzz[iptr] = (c13*xix ) * DxVx + ( c13*xiy ) * DxVy + ( c33*xiz) * DxVz
               + (c13*etx ) * DyVx + ( c13*ety ) * DyVy + ( c33*etz) * DyVz
               + (c13*ztx ) * DzVx + ( c13*zty ) * DzVy + ( c33*ztz) * DzVz;
  

    hTyz[iptr] = ( c55*xiz) * DxVy + ( c55*xiy ) * DxVz
               + ( c55*etz) * DyVy + ( c55*ety ) * DyVz
               + ( c55*ztz) * DzVy + ( c55*zty ) * DzVz;
  
    hTxz[iptr] = ( c55*xiz) * DxVx + (c55*xix ) * DxVz
               + ( c55*etz) * DyVx + (c55*etx ) * DyVz
               + ( c55*ztz) * DzVx + (c55*ztx ) * DzVz;
  
    hTxy[iptr] = ( c66*xiy ) * DxVx + (c66*xix ) * DxVy
               + ( c66*ety ) * DyVx + (c66*etx ) * DyVy
               + ( c66*zty ) * DzVx + (c66*ztx ) * DzVy;

  }

  return;
}

/*******************************************************************************
 * free surface boundary
 ******************************************************************************/

/*
 * implement vlow boundary
 */
__global__ void
sv_eq1st_curv_col_el_vti_rhs_vlow_z2_gpu(
    float *  Vx , float *  Vy , float *  Vz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float *c11d, float *c13d, float *c33d,
    float *c55d, float *c66d, float *slw3d,
    float * matVx2Vz, float * matVy2Vz,
    int ni1, int ni, int nj1, int nj, int nk1, int nk2,
    size_t siz_line, size_t siz_slice,
    int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
    int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
    int num_of_fdz_op, int fdz_max_len, int * fdz_len,
    float *lfdz_coef_all, size_t *lfdz_shift_all,
    const int myid, const int verbose)
{
  // local var
  int k;
  int n_fd; // loop var for fd
  int lfdz_len;
  // local var
  float DxVx,DxVy,DxVz;
  float DyVx,DyVy,DyVz;
  float DzVx,DzVy,DzVz;
  float c11,c13,c33,c55,c66,c12,slw;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;

  float lfdz_coef[5] = {0.0};
  int   lfdz_shift[5] = {0};
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;

  // loop near surface layers
  for (int n=0; n < num_of_fdz_op-1; n++)
  {
    // conver to k index, from surface to inner
    k = nk2 - n;
    // get pos and len for this point
    lfdz_len  = fdz_len[n];
    for (n_fd = 0; n_fd < lfdz_len ; n_fd++) {
      lfdz_shift[n_fd] = lfdz_shift_all[n*fdz_max_len+n_fd];
      lfdz_coef [n_fd]  = lfdz_coef_all [n*fdz_max_len+n_fd];
    }

    if(ix<ni && iy<nj)
    {
      size_t iptr   = (ix+ni1) + (iy+nj1) * siz_line + k * siz_slice;

      // metric
      xix = xi_x[iptr];
      xiy = xi_y[iptr];
      xiz = xi_z[iptr];
      etx = et_x[iptr];
      ety = et_y[iptr];
      etz = et_z[iptr];
      ztx = zt_x[iptr];
      zty = zt_y[iptr];
      ztz = zt_z[iptr];

      // medium
      slw = slw3d[iptr];
      c11 = c11d[iptr];
      c13 = c13d[iptr];
      c33 = c33d[iptr];
      c55 = c55d[iptr];
      c66 = c66d[iptr];
      c12 = c11 - 2.0 * c66;

      // Vx derivatives
      M_FD_SHIFT(DxVx, Vx, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DyVx, Vx, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      // Vy derivatives
      M_FD_SHIFT(DxVy, Vy, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DyVy, Vy, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      // Vz derivatives
      M_FD_SHIFT(DxVz, Vz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DyVz, Vz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      if (k==nk2) // at surface, convert
      {
        size_t ij = ((ix+ni1) + (iy+nj1) * siz_line)*9;
        DzVx = matVx2Vz[ij+3*0+0] * DxVx
             + matVx2Vz[ij+3*0+1] * DxVy
             + matVx2Vz[ij+3*0+2] * DxVz
             + matVy2Vz[ij+3*0+0] * DyVx
             + matVy2Vz[ij+3*0+1] * DyVy
             + matVy2Vz[ij+3*0+2] * DyVz;

        DzVy = matVx2Vz[ij+3*1+0] * DxVx
             + matVx2Vz[ij+3*1+1] * DxVy
             + matVx2Vz[ij+3*1+2] * DxVz
             + matVy2Vz[ij+3*1+0] * DyVx
             + matVy2Vz[ij+3*1+1] * DyVy
             + matVy2Vz[ij+3*1+2] * DyVz;

        DzVz = matVx2Vz[ij+3*2+0] * DxVx
             + matVx2Vz[ij+3*2+1] * DxVy
             + matVx2Vz[ij+3*2+2] * DxVz
             + matVy2Vz[ij+3*2+0] * DyVx
             + matVy2Vz[ij+3*2+1] * DyVy
             + matVy2Vz[ij+3*2+2] * DyVz;
      }
      else // lower than surface, lower order
      {
        M_FD_SHIFT(DzVx, Vx, iptr, lfdz_len, lfdz_shift, lfdz_coef, n_fd);
        M_FD_SHIFT(DzVy, Vy, iptr, lfdz_len, lfdz_shift, lfdz_coef, n_fd);
        M_FD_SHIFT(DzVz, Vz, iptr, lfdz_len, lfdz_shift, lfdz_coef, n_fd);
      }

      // Hooke's equatoin
	    hTxx[iptr] = (c11*xix ) * DxVx + ( c12*xiy ) * DxVy  + ( c13*xiz) * DxVz
                 + (c11*etx ) * DyVx + ( c12*ety ) * DyVy  + ( c13*etz) * DyVz
                 + (c11*ztx ) * DzVx + ( c12*zty ) * DzVy  + ( c13*ztz) * DzVz;
      
      hTyy[iptr] = (c12*xix ) * DxVx + ( c11*xiy ) * DxVy + ( c13*xiz) * DxVz
                 + (c12*etx ) * DyVx + ( c11*ety ) * DyVy + ( c13*etz) * DyVz
                 + (c12*ztx ) * DzVx + ( c11*zty ) * DzVy + ( c13*ztz) * DzVz;
     
      hTzz[iptr] = (c13*xix ) * DxVx + ( c13*xiy ) * DxVy + ( c33*xiz) * DxVz
                 + (c13*etx ) * DyVx + ( c13*ety ) * DyVy + ( c33*etz) * DyVz
                 + (c13*ztx ) * DzVx + ( c13*zty ) * DzVy + ( c33*ztz) * DzVz;
  

      hTyz[iptr] = ( c55*xiz) * DxVy + ( c55*xiy ) * DxVz
                 + ( c55*etz) * DyVy + ( c55*ety ) * DyVz
                 + ( c55*ztz) * DzVy + ( c55*zty ) * DzVz;
  
      hTxz[iptr] = ( c55*xiz) * DxVx + (c55*xix ) * DxVz
                 + ( c55*etz) * DyVx + (c55*etx ) * DyVz
                 + ( c55*ztz) * DzVx + (c55*ztx ) * DzVz;
  
      hTxy[iptr] = ( c66*xiy ) * DxVx + (c66*xix ) * DxVy
                 + ( c66*ety ) * DyVx + (c66*etx ) * DyVy
                 + ( c66*zty ) * DzVx + (c66*ztx ) * DzVy;

    }
  }

  return;
}

/*******************************************************************************
 * CFS-PML boundary
 ******************************************************************************/

/*
 * cfspml, reference to each pml var inside function
 */
void
sv_eq1st_curv_col_el_vti_rhs_cfspml(
    float *  Vx , float *  Vy , float *  Vz ,
    float *  Txx, float *  Tyy, float *  Tzz,
    float *  Txz, float *  Tyz, float *  Txy,
    float * hVx , float * hVy , float * hVz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float *c11d, float *c13d, float *c33d,
    float *c55d, float *c66d, float *slw3d,
    int nk2, size_t siz_line, size_t siz_slice,
    int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
    int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
    int fdz_len, size_t * lfdz_shift, float * lfdz_coef,
    bdrypml_t bdrypml, bdryfree_t bdryfree,
    const int myid, const int verbose)
{
  // check each side
  for (int idim=0; idim<CONST_NDIM; idim++)
  {
    for (int iside=0; iside<2; iside++)
    {
      // skip to next face if not cfspml
      if (bdrypml.is_at_sides[idim][iside] == 0) continue;

      // get index into local var
      int abs_ni1 = bdrypml.ni1[idim][iside];
      int abs_ni2 = bdrypml.ni2[idim][iside];
      int abs_nj1 = bdrypml.nj1[idim][iside];
      int abs_nj2 = bdrypml.nj2[idim][iside];
      int abs_nk1 = bdrypml.nk1[idim][iside];
      int abs_nk2 = bdrypml.nk2[idim][iside];

      
      int abs_ni = abs_ni2-abs_ni1+1; 
      int abs_nj = abs_nj2-abs_nj1+1; 
      int abs_nk = abs_nk2-abs_nk1+1; 
      {
        dim3 block(8,4,4);
        dim3 grid;
        grid.x = (abs_ni+block.x-1)/block.x;
        grid.y = (abs_nj+block.y-1)/block.y;
        grid.z = (abs_nk+block.z-1)/block.z;

        sv_eq1st_curv_col_el_vti_rhs_cfspml_gpu <<<grid, block>>> (
                                idim, iside, Vx , Vy , Vz , 
                                Txx, Tyy, Tzz, Txz, Tyz, Txy, 
                                hVx, hVy, hVz, hTxx, hTyy, hTzz, 
                                hTxz, hTyz, hTxy, xi_x, xi_y, xi_z,
                                et_x, et_y, et_z, zt_x, zt_y, zt_z, 
                                c11d, c13d, c33d, c55d, c66d, slw3d,
                                nk2, siz_line, siz_slice,
                                fdx_len, lfdx_shift,  lfdx_coef,
                                fdy_len, lfdy_shift,  lfdy_coef,
                                fdz_len, lfdz_shift,  lfdz_coef,
                                bdrypml, bdryfree, myid, verbose);
        //hipDeviceSynchronize();
      }
    } // iside
  } // idim

  return;
}

__global__ void
sv_eq1st_curv_col_el_vti_rhs_cfspml_gpu(int idim, int iside,
                                        float *  Vx , float *  Vy , float *  Vz ,
                                        float *  Txx, float *  Tyy, float *  Tzz,
                                        float *  Txz, float *  Tyz, float *  Txy,
                                        float * hVx , float * hVy , float * hVz ,
                                        float * hTxx, float * hTyy, float * hTzz,
                                        float * hTxz, float * hTyz, float * hTxy,
                                        float * xi_x, float * xi_y, float * xi_z,
                                        float * et_x, float * et_y, float * et_z,
                                        float * zt_x, float * zt_y, float * zt_z,
                                        float *c11d, float *c13d, float *c33d,
                                        float *c55d, float *c66d, float *slw3d,
                                        int nk2, size_t siz_line, size_t siz_slice,
                                        int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
                                        int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
                                        int fdz_len, size_t * lfdz_shift, float * lfdz_coef,
                                        bdrypml_t bdrypml, bdryfree_t bdryfree,
                                        const int myid, const int verbose)
{
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  size_t iz = blockIdx.z * blockDim.z + threadIdx.z;
  float *matVx2Vz = bdryfree.matVx2Vz2;
  float *matVy2Vz = bdryfree.matVy2Vz2;
  // local
  size_t iptr, iptr_a;
  float coef_A, coef_B, coef_D, coef_B_minus_1;
  // loop var for fd
  int n_fd;

  // get index into local var
  int abs_ni1 = bdrypml.ni1[idim][iside];
  int abs_ni2 = bdrypml.ni2[idim][iside];
  int abs_nj1 = bdrypml.nj1[idim][iside];
  int abs_nj2 = bdrypml.nj2[idim][iside];
  int abs_nk1 = bdrypml.nk1[idim][iside];
  int abs_nk2 = bdrypml.nk2[idim][iside];

  
  int abs_ni = abs_ni2-abs_ni1+1; 
  int abs_nj = abs_nj2-abs_nj1+1; 
  int abs_nk = abs_nk2-abs_nk1+1; 

  // val on point
  float DxTxx,DxTyy,DxTzz,DxTxy,DxTxz,DxTyz,DxVx,DxVy,DxVz;
  float DyTxx,DyTyy,DyTzz,DyTxy,DyTxz,DyTyz,DyVx,DyVy,DyVz;
  float DzTxx,DzTyy,DzTzz,DzTxy,DzTxz,DzTyz,DzVx,DzVy,DzVz;
  float c11,c13,c33,c55,c66,c12,slw;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;
  float hVx_rhs,hVy_rhs,hVz_rhs;
  float hTxx_rhs,hTyy_rhs,hTzz_rhs,hTxz_rhs,hTyz_rhs,hTxy_rhs;
  // for free surface
  float Dx_DzVx,Dy_DzVx,Dx_DzVy,Dy_DzVy,Dx_DzVz,Dy_DzVz;
  // get coef for this face
  float * ptr_coef_A = bdrypml.A[idim][iside];
  float * ptr_coef_B = bdrypml.B[idim][iside];
  float * ptr_coef_D = bdrypml.D[idim][iside];

  bdrypml_auxvar_t *auxvar = &(bdrypml.auxvar[idim][iside]);

  // get pml vars
  float * abs_vars_cur = auxvar->cur;
  float * abs_vars_rhs = auxvar->rhs;

  float * pml_Vx   = abs_vars_cur + auxvar->Vx_pos;
  float * pml_Vy   = abs_vars_cur + auxvar->Vy_pos;
  float * pml_Vz   = abs_vars_cur + auxvar->Vz_pos;
  float * pml_Txx  = abs_vars_cur + auxvar->Txx_pos;
  float * pml_Tyy  = abs_vars_cur + auxvar->Tyy_pos;
  float * pml_Tzz  = abs_vars_cur + auxvar->Tzz_pos;
  float * pml_Txz  = abs_vars_cur + auxvar->Txz_pos;
  float * pml_Tyz  = abs_vars_cur + auxvar->Tyz_pos;
  float * pml_Txy  = abs_vars_cur + auxvar->Txy_pos;

  float * pml_hVx  = abs_vars_rhs + auxvar->Vx_pos;
  float * pml_hVy  = abs_vars_rhs + auxvar->Vy_pos;
  float * pml_hVz  = abs_vars_rhs + auxvar->Vz_pos;
  float * pml_hTxx = abs_vars_rhs + auxvar->Txx_pos;
  float * pml_hTyy = abs_vars_rhs + auxvar->Tyy_pos;
  float * pml_hTzz = abs_vars_rhs + auxvar->Tzz_pos;
  float * pml_hTxz = abs_vars_rhs + auxvar->Txz_pos;
  float * pml_hTyz = abs_vars_rhs + auxvar->Tyz_pos;
  float * pml_hTxy = abs_vars_rhs + auxvar->Txy_pos;

  // for each dim
  if (idim == 0 ) // x direction
  {
    if(ix<abs_ni  && iy<abs_nj && iz<abs_nk)
    {
      iptr_a = iz*(abs_nj*abs_ni) + iy*abs_ni + ix;
      iptr   = (ix + abs_ni1) + (iy+abs_nj1) * siz_line + (iz+abs_nk1) * siz_slice;
      // pml coefs
      // int abs_i = ix;
      coef_D = ptr_coef_D[ix];
      coef_A = ptr_coef_A[ix];
      coef_B = ptr_coef_B[ix];
      coef_B_minus_1 = coef_B - 1.0;

      // metric
      xix = xi_x[iptr];
      xiy = xi_y[iptr];
      xiz = xi_z[iptr];

      // medium
      slw = slw3d[iptr];
      c11 = c11d[iptr];
      c13 = c13d[iptr];
      c33 = c33d[iptr];
      c55 = c55d[iptr];
      c66 = c66d[iptr];
      c12 = c11 - 2.0 * c66;

      // xi derivatives
      M_FD_SHIFT(DxVx , Vx , iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxVy , Vy , iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxVz , Vz , iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTxx, Txx, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTyy, Tyy, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTzz, Tzz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTxz, Txz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTyz, Tyz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTxy, Txy, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);

      // combine for corr and aux vars
       hVx_rhs = slw * ( xix*DxTxx + xiy*DxTxy + xiz*DxTxz );
       hVy_rhs = slw * ( xix*DxTxy + xiy*DxTyy + xiz*DxTyz );
       hVz_rhs = slw * ( xix*DxTxz + xiy*DxTyz + xiz*DxTzz );
      hTxx_rhs = (c11*xix)*DxVx + (c12*xiy)*DxVy + (c13*xiz)*DxVz; 
      hTyy_rhs = (c12*xix)*DxVx + (c11*xiy)*DxVy + (c13*xiz)*DxVz;
      hTzz_rhs = (c13*xix)*DxVx + (c13*xiy)*DxVy + (c33*xiz)*DxVz;
      hTyz_rhs = (c55*xiz)*DxVy + (c55*xiy)*DxVz;
      hTxz_rhs = (c55*xiz)*DxVx + (c55*xix)*DxVz;
      hTxy_rhs = (c66*xiy)*DxVx + (c66*xix)*DxVy;

      // 1: make corr to moment equation
      hVx[iptr] += coef_B_minus_1 * hVx_rhs - coef_B * pml_Vx[iptr_a];
      hVy[iptr] += coef_B_minus_1 * hVy_rhs - coef_B * pml_Vy[iptr_a];
      hVz[iptr] += coef_B_minus_1 * hVz_rhs - coef_B * pml_Vz[iptr_a];

      // make corr to Hooke's equatoin
      hTxx[iptr] += coef_B_minus_1 * hTxx_rhs - coef_B * pml_Txx[iptr_a];
      hTyy[iptr] += coef_B_minus_1 * hTyy_rhs - coef_B * pml_Tyy[iptr_a];
      hTzz[iptr] += coef_B_minus_1 * hTzz_rhs - coef_B * pml_Tzz[iptr_a];
      hTxz[iptr] += coef_B_minus_1 * hTxz_rhs - coef_B * pml_Txz[iptr_a];
      hTyz[iptr] += coef_B_minus_1 * hTyz_rhs - coef_B * pml_Tyz[iptr_a];
      hTxy[iptr] += coef_B_minus_1 * hTxy_rhs - coef_B * pml_Txy[iptr_a];
      
      // 2: aux var
      //   a1 = alpha + d / beta, dealt in abs_set_cfspml
      pml_hVx[iptr_a]  = coef_D * hVx_rhs  - coef_A * pml_Vx[iptr_a];
      pml_hVy[iptr_a]  = coef_D * hVy_rhs  - coef_A * pml_Vy[iptr_a];
      pml_hVz[iptr_a]  = coef_D * hVz_rhs  - coef_A * pml_Vz[iptr_a];
      pml_hTxx[iptr_a] = coef_D * hTxx_rhs - coef_A * pml_Txx[iptr_a];
      pml_hTyy[iptr_a] = coef_D * hTyy_rhs - coef_A * pml_Tyy[iptr_a];
      pml_hTzz[iptr_a] = coef_D * hTzz_rhs - coef_A * pml_Tzz[iptr_a];
      pml_hTxz[iptr_a] = coef_D * hTxz_rhs - coef_A * pml_Txz[iptr_a];
      pml_hTyz[iptr_a] = coef_D * hTyz_rhs - coef_A * pml_Tyz[iptr_a];
      pml_hTxy[iptr_a] = coef_D * hTxy_rhs - coef_A * pml_Txy[iptr_a];

      // add contributions from free surface condition
      //  not consider timg because conflict with main cfspml,
      //     need to revise in the future if required
      if (bdryfree.is_at_sides[CONST_NDIM-1][1]==1 && (iz+abs_nk1)==nk2)
      {
        // zeta derivatives
        size_t ij = ((ix+abs_ni1) + (iy+abs_nj1) * siz_line)*9;
        Dx_DzVx = matVx2Vz[ij+3*0+0] * DxVx
                + matVx2Vz[ij+3*0+1] * DxVy
                + matVx2Vz[ij+3*0+2] * DxVz;

        Dx_DzVy = matVx2Vz[ij+3*1+0] * DxVx
                + matVx2Vz[ij+3*1+1] * DxVy
                + matVx2Vz[ij+3*1+2] * DxVz;

        Dx_DzVz = matVx2Vz[ij+3*2+0] * DxVx
                + matVx2Vz[ij+3*2+1] * DxVy
                + matVx2Vz[ij+3*2+2] * DxVz;

        // metric
        ztx = zt_x[iptr];
        zty = zt_y[iptr];
        ztz = zt_z[iptr];

        // keep xi derivative terms, including free surface convered
        hTxx_rhs = (c11*ztx)*Dx_DzVx + (c12*zty)*Dx_DzVy + (c13*ztz)*Dx_DzVz; 
        hTyy_rhs = (c12*ztx)*Dx_DzVx + (c11*zty)*Dx_DzVy + (c13*ztz)*Dx_DzVz;
        hTzz_rhs = (c13*ztx)*Dx_DzVx + (c13*zty)*Dx_DzVy + (c33*ztz)*Dx_DzVz;
        hTyz_rhs = (c55*ztz)*Dx_DzVy + (c55*zty)*Dx_DzVz;
        hTxz_rhs = (c55*ztz)*Dx_DzVx + (c55*ztx)*Dx_DzVz;
        hTxy_rhs = (c66*zty)*Dx_DzVx + (c66*ztx)*Dx_DzVy;

        // make corr to Hooke's equatoin
        hTxx[iptr] += (coef_B - 1.0) * hTxx_rhs;
        hTyy[iptr] += (coef_B - 1.0) * hTyy_rhs;
        hTzz[iptr] += (coef_B - 1.0) * hTzz_rhs;
        hTxz[iptr] += (coef_B - 1.0) * hTxz_rhs;
        hTyz[iptr] += (coef_B - 1.0) * hTyz_rhs;
        hTxy[iptr] += (coef_B - 1.0) * hTxy_rhs;

        // aux var
        //   a1 = alpha + d / beta, dealt in abs_set_cfspml
        pml_hTxx[iptr_a] += coef_D * hTxx_rhs;
        pml_hTyy[iptr_a] += coef_D * hTyy_rhs;
        pml_hTzz[iptr_a] += coef_D * hTzz_rhs;
        pml_hTxz[iptr_a] += coef_D * hTxz_rhs;
        pml_hTyz[iptr_a] += coef_D * hTyz_rhs;
        pml_hTxy[iptr_a] += coef_D * hTxy_rhs;
      } 
    }
  }
  else if (idim == 1) // y direction
  {
    if(ix<abs_ni  && iy<abs_nj && iz<abs_nk)
    {
      iptr_a = iz*(abs_nj*abs_ni) + iy*abs_ni + ix;
      iptr   = (ix + abs_ni1) + (iy+abs_nj1)*siz_line + (iz+abs_nk1) * siz_slice;

      // pml coefs
      // int abs_j = iy;
      coef_D = ptr_coef_D[iy];
      coef_A = ptr_coef_A[iy];
      coef_B = ptr_coef_B[iy];
      coef_B_minus_1 = coef_B - 1.0;

      // metric
      etx = et_x[iptr];
      ety = et_y[iptr];
      etz = et_z[iptr];

      // medium
      slw = slw3d[iptr];
      c11 = c11d[iptr];
      c13 = c13d[iptr];
      c33 = c33d[iptr];
      c55 = c55d[iptr];
      c66 = c66d[iptr];
      c12 = c11 - 2.0 * c66;

      // et derivatives
      M_FD_SHIFT(DyVx , Vx , iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyVy , Vy , iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyVz , Vz , iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTxx, Txx, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTyy, Tyy, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTzz, Tzz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTxz, Txz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTyz, Tyz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTxy, Txy, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      // combine for corr and aux vars
       hVx_rhs = slw * ( etx*DyTxx + ety*DyTxy + etz*DyTxz );
       hVy_rhs = slw * ( etx*DyTxy + ety*DyTyy + etz*DyTyz );
       hVz_rhs = slw * ( etx*DyTxz + ety*DyTyz + etz*DyTzz );
      hTxx_rhs = (c11*etx)*DyVx + (c12*ety)*DyVy + (c13*etz)*DyVz; 
      hTyy_rhs = (c12*etx)*DyVx + (c11*ety)*DyVy + (c13*etz)*DyVz;
      hTzz_rhs = (c13*etx)*DyVx + (c13*ety)*DyVy + (c33*etz)*DyVz;
      hTyz_rhs = (c55*etz)*DyVy + (c55*ety)*DyVz;
      hTxz_rhs = (c55*etz)*DyVx + (c55*etx)*DyVz;
      hTxy_rhs = (c66*ety)*DyVx + (c66*etx)*DyVy;

      // 1: make corr to moment equation
      hVx[iptr] += coef_B_minus_1 * hVx_rhs - coef_B * pml_Vx[iptr_a];
      hVy[iptr] += coef_B_minus_1 * hVy_rhs - coef_B * pml_Vy[iptr_a];
      hVz[iptr] += coef_B_minus_1 * hVz_rhs - coef_B * pml_Vz[iptr_a];

      // make corr to Hooke's equatoin
      hTxx[iptr] += coef_B_minus_1 * hTxx_rhs - coef_B * pml_Txx[iptr_a];
      hTyy[iptr] += coef_B_minus_1 * hTyy_rhs - coef_B * pml_Tyy[iptr_a];
      hTzz[iptr] += coef_B_minus_1 * hTzz_rhs - coef_B * pml_Tzz[iptr_a];
      hTxz[iptr] += coef_B_minus_1 * hTxz_rhs - coef_B * pml_Txz[iptr_a];
      hTyz[iptr] += coef_B_minus_1 * hTyz_rhs - coef_B * pml_Tyz[iptr_a];
      hTxy[iptr] += coef_B_minus_1 * hTxy_rhs - coef_B * pml_Txy[iptr_a];
      
      // 2: aux var
      //   a1 = alpha + d / beta, dealt in abs_set_cfspml
      pml_hVx[iptr_a]  = coef_D * hVx_rhs  - coef_A * pml_Vx[iptr_a];
      pml_hVy[iptr_a]  = coef_D * hVy_rhs  - coef_A * pml_Vy[iptr_a];
      pml_hVz[iptr_a]  = coef_D * hVz_rhs  - coef_A * pml_Vz[iptr_a];
      pml_hTxx[iptr_a] = coef_D * hTxx_rhs - coef_A * pml_Txx[iptr_a];
      pml_hTyy[iptr_a] = coef_D * hTyy_rhs - coef_A * pml_Tyy[iptr_a];
      pml_hTzz[iptr_a] = coef_D * hTzz_rhs - coef_A * pml_Tzz[iptr_a];
      pml_hTxz[iptr_a] = coef_D * hTxz_rhs - coef_A * pml_Txz[iptr_a];
      pml_hTyz[iptr_a] = coef_D * hTyz_rhs - coef_A * pml_Tyz[iptr_a];
      pml_hTxy[iptr_a] = coef_D * hTxy_rhs - coef_A * pml_Txy[iptr_a];

      // add contributions from free surface condition
      if (bdryfree.is_at_sides[CONST_NDIM-1][1]==1 && (iz+abs_nk1)==nk2)
      {
        // zeta derivatives
        size_t ij = ((ix+abs_ni1) + (iy+abs_nj1) * siz_line)*9;
        Dy_DzVx = matVy2Vz[ij+3*0+0] * DyVx
                + matVy2Vz[ij+3*0+1] * DyVy
                + matVy2Vz[ij+3*0+2] * DyVz;

        Dy_DzVy = matVy2Vz[ij+3*1+0] * DyVx
                + matVy2Vz[ij+3*1+1] * DyVy
                + matVy2Vz[ij+3*1+2] * DyVz;

        Dy_DzVz = matVy2Vz[ij+3*2+0] * DyVx
                + matVy2Vz[ij+3*2+1] * DyVy
                + matVy2Vz[ij+3*2+2] * DyVz;

        // metric
        ztx = zt_x[iptr];
        zty = zt_y[iptr];
        ztz = zt_z[iptr];

        // keep eta derivative terms, including free surface convered
        hTxx_rhs = (c11*ztx)*Dy_DzVx + (c12*zty)*Dy_DzVy + (c13*ztz)*Dy_DzVz; 
        hTyy_rhs = (c12*ztx)*Dy_DzVx + (c11*zty)*Dy_DzVy + (c13*ztz)*Dy_DzVz;
        hTzz_rhs = (c13*ztx)*Dy_DzVx + (c13*zty)*Dy_DzVy + (c33*ztz)*Dy_DzVz;
        hTyz_rhs = (c55*ztz)*Dy_DzVy + (c55*zty)*Dy_DzVz;
        hTxz_rhs = (c55*ztz)*Dy_DzVx + (c55*ztx)*Dy_DzVz;
        hTxy_rhs = (c66*zty)*Dy_DzVx + (c66*ztx)*Dy_DzVy;

        // make corr to Hooke's equatoin
        hTxx[iptr] += (coef_B - 1.0) * hTxx_rhs;
        hTyy[iptr] += (coef_B - 1.0) * hTyy_rhs;
        hTzz[iptr] += (coef_B - 1.0) * hTzz_rhs;
        hTxz[iptr] += (coef_B - 1.0) * hTxz_rhs;
        hTyz[iptr] += (coef_B - 1.0) * hTyz_rhs;
        hTxy[iptr] += (coef_B - 1.0) * hTxy_rhs;

        // aux var
        //   a1 = alpha + d / beta, dealt in abs_set_cfspml
        pml_hTxx[iptr_a] += coef_D * hTxx_rhs;
        pml_hTyy[iptr_a] += coef_D * hTyy_rhs;
        pml_hTzz[iptr_a] += coef_D * hTzz_rhs;
        pml_hTxz[iptr_a] += coef_D * hTxz_rhs;
        pml_hTyz[iptr_a] += coef_D * hTyz_rhs;
        pml_hTxy[iptr_a] += coef_D * hTxy_rhs;
      }
    }
  }
  else // z direction
  {
    if(ix<abs_ni  && iy<abs_nj && iz<abs_nk)
    {
      iptr_a = iz*(abs_nj*abs_ni) + iy*abs_ni + ix;
      iptr   = (ix + abs_ni1) + (iy+abs_nj1) * siz_line + (iz+abs_nk1) * siz_slice;
      // pml coefs
      // int abs_k = iz;
      coef_D = ptr_coef_D[iz];
      coef_A = ptr_coef_A[iz];
      coef_B = ptr_coef_B[iz];
      coef_B_minus_1 = coef_B - 1.0;

      // metric
      ztx = zt_x[iptr];
      zty = zt_y[iptr];
      ztz = zt_z[iptr];

      // medium
      slw = slw3d[iptr];
      c11 = c11d[iptr];
      c13 = c13d[iptr];
      c33 = c33d[iptr];
      c55 = c55d[iptr];
      c66 = c66d[iptr];
      c12 = c11 - 2.0 * c66;

      // zt derivatives
      M_FD_SHIFT(DzVx , Vx , iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzVy , Vy , iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzVz , Vz , iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTxx, Txx, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTyy, Tyy, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTzz, Tzz, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTxz, Txz, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTyz, Tyz, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTxy, Txy, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);

      // combine for corr and aux vars
       hVx_rhs = slw * ( ztx*DzTxx + zty*DzTxy + ztz*DzTxz );
       hVy_rhs = slw * ( ztx*DzTxy + zty*DzTyy + ztz*DzTyz );
       hVz_rhs = slw * ( ztx*DzTxz + zty*DzTyz + ztz*DzTzz );
      hTxx_rhs = (c11*ztx)*DzVx + (c12*zty)*DzVy + (c13*ztz)*DzVz; 
      hTyy_rhs = (c12*ztx)*DzVx + (c11*zty)*DzVy + (c13*ztz)*DzVz;
      hTzz_rhs = (c13*ztx)*DzVx + (c13*zty)*DzVy + (c33*ztz)*DzVz;
      hTyz_rhs = (c55*ztz)*DzVy + (c55*zty)*DzVz;
      hTxz_rhs = (c55*ztz)*DzVx + (c55*ztx)*DzVz;
      hTxy_rhs = (c66*zty)*DzVx + (c66*ztx)*DzVy;

      // 1: make corr to moment equation
      hVx[iptr] += coef_B_minus_1 * hVx_rhs - coef_B * pml_Vx[iptr_a];
      hVy[iptr] += coef_B_minus_1 * hVy_rhs - coef_B * pml_Vy[iptr_a];
      hVz[iptr] += coef_B_minus_1 * hVz_rhs - coef_B * pml_Vz[iptr_a];

      // make corr to Hooke's equatoin
      hTxx[iptr] += coef_B_minus_1 * hTxx_rhs - coef_B * pml_Txx[iptr_a];
      hTyy[iptr] += coef_B_minus_1 * hTyy_rhs - coef_B * pml_Tyy[iptr_a];
      hTzz[iptr] += coef_B_minus_1 * hTzz_rhs - coef_B * pml_Tzz[iptr_a];
      hTxz[iptr] += coef_B_minus_1 * hTxz_rhs - coef_B * pml_Txz[iptr_a];
      hTyz[iptr] += coef_B_minus_1 * hTyz_rhs - coef_B * pml_Tyz[iptr_a];
      hTxy[iptr] += coef_B_minus_1 * hTxy_rhs - coef_B * pml_Txy[iptr_a];
      
      // 2: aux var
      //   a1 = alpha + d / beta, dealt in abs_set_cfspml
      pml_hVx[iptr_a]  = coef_D * hVx_rhs  - coef_A * pml_Vx[iptr_a];
      pml_hVy[iptr_a]  = coef_D * hVy_rhs  - coef_A * pml_Vy[iptr_a];
      pml_hVz[iptr_a]  = coef_D * hVz_rhs  - coef_A * pml_Vz[iptr_a];
      pml_hTxx[iptr_a] = coef_D * hTxx_rhs - coef_A * pml_Txx[iptr_a];
      pml_hTyy[iptr_a] = coef_D * hTyy_rhs - coef_A * pml_Tyy[iptr_a];
      pml_hTzz[iptr_a] = coef_D * hTzz_rhs - coef_A * pml_Tzz[iptr_a];
      pml_hTxz[iptr_a] = coef_D * hTxz_rhs - coef_A * pml_Txz[iptr_a];
      pml_hTyz[iptr_a] = coef_D * hTyz_rhs - coef_A * pml_Tyz[iptr_a];
      pml_hTxy[iptr_a] = coef_D * hTxy_rhs - coef_A * pml_Txy[iptr_a];
    } 
  } 

  return;
}

/*******************************************************************************
 * free surface coef
 * converted matrix for velocity gradient
 *  only implement z2 (top) right now
 ******************************************************************************/

__global__ void
sv_eq1st_curv_col_el_vti_dvh2dvz_gpu(gdinfo_t        gdinfo_d,
                                     gdcurv_metric_t metric_d,
                                     md_t       md_d,
                                     bdryfree_t      bdryfree_d,
                                     const int verbose)
{
  int ni1 = gdinfo_d.ni1;
  int ni2 = gdinfo_d.ni2;
  int nj1 = gdinfo_d.nj1;
  int nj2 = gdinfo_d.nj2;
  int nk1 = gdinfo_d.nk1;
  int nk2 = gdinfo_d.nk2;
  int nx  = gdinfo_d.nx;
  int ny  = gdinfo_d.ny;
  int nz  = gdinfo_d.nz;
  size_t siz_line   = gdinfo_d.siz_iy;
  size_t siz_slice  = gdinfo_d.siz_iz;
  size_t siz_volume = gdinfo_d.siz_icmp;

  // point to each var
  float * xi_x = metric_d.xi_x;
  float * xi_y = metric_d.xi_y;
  float * xi_z = metric_d.xi_z;
  float * et_x = metric_d.eta_x;
  float * et_y = metric_d.eta_y;
  float * et_z = metric_d.eta_z;
  float * zt_x = metric_d.zeta_x;
  float * zt_y = metric_d.zeta_y;
  float * zt_z = metric_d.zeta_z;

  float * c11d = md_d.c11;
  float * c13d = md_d.c13;
  float * c33d = md_d.c33;
  float * c55d = md_d.c55;
  float * c66d = md_d.c66;

  float *matVx2Vz = bdryfree_d.matVx2Vz2;
  float *matVy2Vz = bdryfree_d.matVy2Vz2;

  float A[3][3], B[3][3], C[3][3];
  float AB[3][3], AC[3][3];

  float c11,c13,c33,c55,c66,c12;
  float xix, xiy ,xiz, etx, ety, etz, ztx, zty, ztz;
 
  int k = nk2;

  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  if(ix<(ni2-ni1+1) && iy<(nj2-nj1+1))
  {
    size_t iptr = (ix+ni1) + (iy+nj1) * siz_line + k * siz_slice;

    xix = xi_x[iptr];
    xiy = xi_y[iptr];
    xiz = xi_z[iptr];
    etx = et_x[iptr];
    ety = et_y[iptr];
    etz = et_z[iptr];
    ztx = zt_x[iptr];
    zty = zt_y[iptr];
    ztz = zt_z[iptr];
    
    c11 = c11d[iptr];
    c13 = c13d[iptr];
    c33 = c33d[iptr];
    c55 = c55d[iptr];
    c66 = c66d[iptr];
    c12 = c11 - 2.0 * c66;

    // first dim: irow; sec dim: jcol, as Fortran code
    A[0][0] = (c11*ztx)*ztx + (c66*zty)*zty + (c55*ztz)*ztz;
    A[0][1] = (c12*zty)*ztx + (c66*ztx)*zty;
    A[0][2] = (c13*ztz)*ztx + (c55*ztx)*ztz; 
    A[1][0] = (c66*zty)*ztx + (c12*ztx)*zty; 
    A[1][1] = (c66*ztx)*ztx + (c11*zty)*zty + (c55*ztz)*ztz; 
    A[1][2] = (c13*ztz)*zty + (c55*zty)*ztz;
    A[2][0] = (c55*ztz)*ztx + (c13*ztx)*ztz;
    A[2][1] = (c55*ztz)*zty + (c13*zty)*ztz;
    A[2][2] = (c55*ztx)*ztx + (c55*zty)*zty + (c33*ztz)*ztz; 
    fdlib_math_invert3x3(A);
                                                     
    B[0][0] = (c11*xix)*ztx + (c66*xiy)*zty + (c55*xiz)*ztz;
    B[0][1] = (c12*xiy)*ztx + (c66*xix)*zty;
    B[0][2] = (c13*xiz)*ztx + (c55*xix)*ztz; 
    B[1][0] = (c66*xiy)*ztx + (c12*xix)*zty; 
    B[1][1] = (c66*xix)*ztx + (c11*xiy)*zty + (c55*xiz)*ztz; 
    B[1][2] = (c13*xiz)*zty + (c55*xiy)*ztz;
    B[2][0] = (c55*xiz)*ztx + (c13*xix)*ztz;
    B[2][1] = (c55*xiz)*zty + (c13*xiy)*ztz;
    B[2][2] = (c55*xix)*ztx + (c55*xiy)*zty + (c33*xiz)*ztz; 
     
    C[0][0] = (c11*etx)*ztx + (c66*ety)*zty + (c55*etz)*ztz;
    C[0][1] = (c12*ety)*ztx + (c66*etx)*zty;
    C[0][2] = (c13*etz)*ztx + (c55*etx)*ztz; 
    C[1][0] = (c66*ety)*ztx + (c12*etx)*zty; 
    C[1][1] = (c66*etx)*ztx + (c11*ety)*zty + (c55*etz)*ztz; 
    C[1][2] = (c13*etz)*zty + (c55*ety)*ztz;
    C[2][0] = (c55*etz)*ztx + (c13*etx)*ztz;
    C[2][1] = (c55*etz)*zty + (c13*ety)*ztz;
    C[2][2] = (c55*etx)*ztx + (c55*ety)*zty + (c33*etz)*ztz; 
    fdlib_math_matmul3x3(A, B, AB);
    fdlib_math_matmul3x3(A, C, AC);

    size_t ij = ((iy+nj1) * siz_line + (ix+ni1)) * 9;

    // save into mat
    for(int irow = 0; irow < 3; irow++){
      for(int jcol = 0; jcol < 3; jcol++){
        matVx2Vz[ij + irow*3 + jcol] = -1.0f * AB[irow][jcol];
        matVy2Vz[ij + irow*3 + jcol] = -1.0f * AC[irow][jcol];
      }
    }
  }

  return;
}

