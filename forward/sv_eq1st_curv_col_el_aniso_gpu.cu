#include "hip/hip_runtime.h"
/*******************************************************************************
 * solver of isotropic elastic 1st-order eqn using curv grid and macdrp schem
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <mpi.h>

#include "fdlib_mem.h"
#include "fdlib_math.h"
#include "sv_eq1st_curv_col_el_iso_gpu.h"
#include "sv_eq1st_curv_col_el_aniso_gpu.h"
#include "cuda_common.h"

//#define SV_EQ1ST_CURV_COLGRD_ISO_DEBUG

/*******************************************************************************
 * perform one stage calculation of rhs
 ******************************************************************************/

void
sv_eq1st_curv_col_el_aniso_onestage(
  float *w_cur_d,
  float *rhs_d, 
  wav_t  wav_d,
  fd_wav_t fd_wav_d,
  gdinfo_t  gdinfo_d,
  gdcurv_metric_t metric_d,
  md_t md_d,
  bdryfree_t bdryfree_d,
  bdrypml_t  bdrypml_d,
  src_t src_d,
  // include different order/stentil
  int num_of_fdx_op, fd_op_t *fdx_op,
  int num_of_fdy_op, fd_op_t *fdy_op,
  int num_of_fdz_op, fd_op_t *fdz_op,
  int fdz_max_len, 
  const int myid, const int verbose)
{
  // local pointer get each vars
  float *Vx    = w_cur_d + wav_d.Vx_pos ;
  float *Vy    = w_cur_d + wav_d.Vy_pos ;
  float *Vz    = w_cur_d + wav_d.Vz_pos ;
  float *Txx   = w_cur_d + wav_d.Txx_pos;
  float *Tyy   = w_cur_d + wav_d.Tyy_pos;
  float *Tzz   = w_cur_d + wav_d.Tzz_pos;
  float *Txz   = w_cur_d + wav_d.Txz_pos;
  float *Tyz   = w_cur_d + wav_d.Tyz_pos;
  float *Txy   = w_cur_d + wav_d.Txy_pos;
  float *hVx   = rhs_d   + wav_d.Vx_pos ; 
  float *hVy   = rhs_d   + wav_d.Vy_pos ; 
  float *hVz   = rhs_d   + wav_d.Vz_pos ; 
  float *hTxx  = rhs_d   + wav_d.Txx_pos; 
  float *hTyy  = rhs_d   + wav_d.Tyy_pos; 
  float *hTzz  = rhs_d   + wav_d.Tzz_pos; 
  float *hTxz  = rhs_d   + wav_d.Txz_pos; 
  float *hTyz  = rhs_d   + wav_d.Tyz_pos; 
  float *hTxy  = rhs_d   + wav_d.Txy_pos; 

  float *xi_x  = metric_d.xi_x;
  float *xi_y  = metric_d.xi_y;
  float *xi_z  = metric_d.xi_z;
  float *et_x  = metric_d.eta_x;
  float *et_y  = metric_d.eta_y;
  float *et_z  = metric_d.eta_z;
  float *zt_x  = metric_d.zeta_x;
  float *zt_y  = metric_d.zeta_y;
  float *zt_z  = metric_d.zeta_z;
  float *jac3d = metric_d.jac;

  float *c11   = md_d.c11;
  float *c12   = md_d.c12;
  float *c13   = md_d.c13;
  float *c14   = md_d.c14;
  float *c15   = md_d.c15;
  float *c16   = md_d.c16;
  float *c22   = md_d.c22;
  float *c23   = md_d.c23;
  float *c24   = md_d.c24;
  float *c25   = md_d.c25;
  float *c26   = md_d.c26;
  float *c33   = md_d.c33;
  float *c34   = md_d.c34;
  float *c35   = md_d.c35;
  float *c36   = md_d.c36;
  float *c44   = md_d.c44;
  float *c45   = md_d.c45;
  float *c46   = md_d.c46;
  float *c55   = md_d.c55;
  float *c56   = md_d.c56;
  float *c66   = md_d.c66;
  float *slw3d = md_d.rho;

  // grid size
  int ni1 = gdinfo_d.ni1;
  int ni2 = gdinfo_d.ni2;
  int nj1 = gdinfo_d.nj1;
  int nj2 = gdinfo_d.nj2;
  int nk1 = gdinfo_d.nk1;
  int nk2 = gdinfo_d.nk2;

  int ni  = gdinfo_d.ni;
  int nj  = gdinfo_d.nj;
  int nk  = gdinfo_d.nk;
  int nx  = gdinfo_d.nx;
  int ny  = gdinfo_d.ny;
  int nz  = gdinfo_d.nz;
  size_t siz_line   = gdinfo_d.siz_line;
  size_t siz_slice  = gdinfo_d.siz_slice;
  size_t siz_volume = gdinfo_d.siz_volume;

  float *matVx2Vz = bdryfree_d.matVx2Vz2;
  float *matVy2Vz = bdryfree_d.matVy2Vz2;

  // local fd op
  int    fdx_len;
  int    *fdx_indx;
  float  *fdx_coef;
  int    fdy_len;
  int    *fdy_indx;
  float  *fdy_coef;
  int    fdz_len;
  int    *fdz_indx;
  float  *fdz_coef;

  // for get a op from 1d array, currently use num_of_fdz_op as index
  // length, index, coef of a op
  fdx_len  = fdx_op[num_of_fdx_op-1].total_len;
  fdx_indx = fdx_op[num_of_fdx_op-1].indx;
  fdx_coef = fdx_op[num_of_fdx_op-1].coef;

  fdy_len  = fdy_op[num_of_fdy_op-1].total_len;
  fdy_indx = fdy_op[num_of_fdy_op-1].indx;
  fdy_coef = fdy_op[num_of_fdy_op-1].coef;

  fdz_len  = fdz_op[num_of_fdz_op-1].total_len;
  fdz_indx = fdz_op[num_of_fdz_op-1].indx;
  fdz_coef = fdz_op[num_of_fdz_op-1].coef;

  // use local stack array for speedup
  float  lfdx_coef [fdx_len];
  size_t lfdx_shift[fdx_len];
  float  lfdy_coef [fdy_len];
  size_t lfdy_shift[fdy_len];
  float  lfdz_coef [fdz_len];
  size_t lfdz_shift[fdz_len];

  // put fd op into local array
  for (int i=0; i < fdx_len; i++) {
    lfdx_coef [i] = fdx_coef[i];
    lfdx_shift[i] = fdx_indx[i];
  }
  for (int j=0; j < fdy_len; j++) {
    lfdy_coef [j] = fdy_coef[j];
    lfdy_shift[j] = fdy_indx[j] * siz_line;
  }
  for (int k=0; k < fdz_len; k++) {
    lfdz_coef [k] = fdz_coef[k];
    lfdz_shift[k] = fdz_indx[k] * siz_slice;
  }

  // allocate max_len because fdz may have different lens
  // these array is for low order surface
  float  fdz_coef_all [num_of_fdz_op*fdz_max_len];
  size_t fdz_shift_all[num_of_fdz_op*fdz_max_len];
  int    fdz_len_all[num_of_fdz_op];
  // loop near surface layers
  for (int n=0; n < num_of_fdz_op-1; n++)
  {
    // get pos and len for this point
    fdz_len_all[n]  = fdz_op[n].total_len;
    // point to indx/coef for this point
    int   *p_fdz_indx  = fdz_op[n].indx;
    float *p_fdz_coef  = fdz_op[n].coef;
    for (int n_fd = 0; n_fd < fdz_len_all[n] ; n_fd++) {
      fdz_shift_all[n_fd + n*fdz_max_len]  = p_fdz_indx[n_fd] * siz_slice;
      fdz_coef_all [n_fd + n*fdz_max_len]  = p_fdz_coef[n_fd];
    }
  }

  int  *lfdz_len_d = fd_wav_d.fdz_len_d;
  float *lfdx_coef_d = fd_wav_d.fdx_coef_d;
  float *lfdy_coef_d = fd_wav_d.fdy_coef_d;
  float *lfdz_coef_d = fd_wav_d.fdz_coef_d;
  float *lfdz_coef_all_d = fd_wav_d.fdz_coef_all_d;
  size_t  *lfdx_shift_d = fd_wav_d.fdx_shift_d;
  size_t  *lfdy_shift_d = fd_wav_d.fdy_shift_d;
  size_t  *lfdz_shift_d = fd_wav_d.fdz_shift_d;
  size_t  *lfdz_shift_all_d = fd_wav_d.fdz_shift_all_d;
  int  *lfdx_indx_d = fd_wav_d.fdx_indx_d;
  int  *lfdy_indx_d = fd_wav_d.fdy_indx_d;
  int  *lfdz_indx_d = fd_wav_d.fdz_indx_d;
  int  *lfdz_indx_all_d = fd_wav_d.fdz_indx_all_d;
  //host to device
  CUDACHECK(hipMemcpy(lfdx_coef_d,lfdx_coef,fdx_len*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdy_coef_d,lfdy_coef,fdy_len*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_coef_d,lfdz_coef,fdz_len*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdx_shift_d,lfdx_shift,fdx_len*sizeof(size_t),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdy_shift_d,lfdy_shift,fdy_len*sizeof(size_t),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_shift_d,lfdz_shift,fdz_len*sizeof(size_t),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdx_indx_d,fdx_indx,fdx_len*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdy_indx_d,fdy_indx,fdy_len*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_indx_d,fdz_indx,fdz_len*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_len_d,fdz_len_all,num_of_fdz_op*sizeof(int),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_coef_all_d,fdz_coef_all,fdz_max_len*num_of_fdz_op*sizeof(float),hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(lfdz_shift_all_d,fdz_shift_all,fdz_max_len*num_of_fdz_op*sizeof(size_t),hipMemcpyHostToDevice));

  {
    dim3 block(8,8,8);
    dim3 grid;
    grid.x = (ni+block.x-1)/block.x;
    grid.y = (nj+block.y-1)/block.y;
    grid.z = (nk+block.z-1)/block.z;
    sv_eq1st_curv_col_el_aniso_rhs_inner_gpu <<<grid, block>>> (
                        Vx,Vy,Vz,Txx,Tyy,Tzz,Txz,Tyz,Txy,
                        hVx,hVy,hVz,hTxx,hTyy,hTzz,hTxz,hTyz,hTxy,
                        xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                        c11,c12,c13,c14,c15,c16,
                            c22,c23,c24,c25,c26,
                                c33,c34,c35,c36,
                                    c44,c45,c46,
                                        c55,c56,
                                            c66, slw3d,
                        ni1,ni,nj1,nj,nk1,nk,siz_line,siz_slice,
                        fdx_len, lfdx_shift_d, lfdx_coef_d,
                        fdy_len, lfdy_shift_d, lfdy_coef_d,
                        fdz_len, lfdz_shift_d, lfdz_coef_d,
                        myid, verbose);
  }

  // free surface at z2
  if (bdryfree_d.is_at_sides[2][1] == 1)
  {
    // tractiong
    {
      dim3 block(8,8);
      dim3 grid;
      grid.x = (ni+block.x-1)/block.x;
      grid.y = (nj+block.y-1)/block.y;
      sv_eq1st_curv_col_el_iso_rhs_timg_z2_gpu  <<<grid, block>>> (
                          Txx,Tyy,Tzz,Txz,Tyz,Txy,hVx,hVy,hVz,
                          xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                          jac3d, slw3d,
                          ni1,ni,nj1,nj,nk1,nk2,siz_line,siz_slice,
                          fdx_len, lfdx_indx_d, lfdx_coef_d,
                          fdy_len, lfdy_indx_d, lfdy_coef_d,
                          fdz_len, lfdz_indx_d, lfdz_coef_d,
                          myid, verbose);
      hipDeviceSynchronize();
    }

    // velocity: vlow
    {
      dim3 block(8,8);
      dim3 grid;
      grid.x = (ni+block.x-1)/block.x;
      grid.y = (nj+block.y-1)/block.y;
      sv_eq1st_curv_col_el_aniso_rhs_vlow_z2_gpu  <<<grid, block>>> (
                        Vx,Vy,Vz,hTxx,hTyy,hTzz,hTxz,hTyz,hTxy,
                        xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                        c11,c12,c13,c14,c15,c16,
                            c22,c23,c24,c25,c26,
                                c33,c34,c35,c36,
                                    c44,c45,c46,
                                        c55,c56,
                                            c66, slw3d,
                        matVx2Vz,matVy2Vz,
                        ni1,ni,nj1,nj,nk1,nk2,siz_line,siz_slice,
                        fdx_len, lfdx_shift_d, lfdx_coef_d,
                        fdy_len, lfdy_shift_d, lfdy_coef_d,
                        num_of_fdz_op,fdz_max_len,lfdz_len_d,
                        lfdz_coef_all_d,lfdz_shift_all_d,
                        myid, verbose);
      CUDACHECK( hipDeviceSynchronize() );
    }
  }

  // cfs-pml, loop face inside
  if (bdrypml_d.is_enable == 1)
  {
    sv_eq1st_curv_col_el_aniso_rhs_cfspml(Vx,Vy,Vz,Txx,Tyy,Tzz,Txz,Tyz,Txy,
                                        hVx,hVy,hVz,hTxx,hTyy,hTzz,hTxz,hTyz,hTxy,
                                        xi_x, xi_y, xi_z, et_x, et_y, et_z, zt_x, zt_y, zt_z,
                                        c11,c12,c13,c14,c15,c16,
                                            c22,c23,c24,c25,c26,
                                                c33,c34,c35,c36,
                                                    c44,c45,c46,
                                                        c55,c56,
                                                            c66, slw3d,
                                        nk2, siz_line,siz_slice,
                                        fdx_len, lfdx_shift_d, lfdx_coef_d,
                                        fdy_len, lfdy_shift_d, lfdy_coef_d,
                                        fdz_len, lfdz_shift_d, lfdz_coef_d,
                                        bdrypml_d, bdryfree_d,
                                        myid, verbose);
  }

  // add source term
  if (src_d.total_number > 0)
  {
    {
      dim3 block(256);
      dim3 grid;
      grid.x = (src_d.total_number+block.x-1) / block.x;
      sv_eq1st_curv_col_el_iso_rhs_src_gpu  <<< grid,block >>> (
                        hVx, hVy, hVz, hTxx, hTyy, hTzz, hTxz, hTyz, hTxy,
                        jac3d, slw3d, 
                        src_d,
                        myid, verbose);
      CUDACHECK( hipDeviceSynchronize() );
    }
  }

  return;
}

/*******************************************************************************
 * calculate all points without boundaries treatment
 ******************************************************************************/

__global__ void
sv_eq1st_curv_col_el_aniso_rhs_inner_gpu(
    float *  Vx , float *  Vy , float *  Vz ,
    float *  Txx, float *  Tyy, float *  Tzz,
    float *  Txz, float *  Tyz, float *  Txy,
    float * hVx , float * hVy , float * hVz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float * c11d, float * c12d, float * c13d,
    float * c14d, float * c15d, float * c16d,
                  float * c22d, float * c23d,
    float * c24d, float * c25d, float * c26d,
                                float * c33d,
    float * c34d, float * c35d, float * c36d,
    float * c44d, float * c45d, float * c46d,
                  float * c55d, float * c56d,
                                float * c66d,
                                float * slw3d,
    int ni1, int ni, int nj1, int nj, int nk1, int nk,
    size_t siz_line, size_t siz_slice,
    int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
    int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
    int fdz_len, size_t * lfdz_shift, float * lfdz_coef,
    const int myid, const int verbose)
{
  // use local stack array for speedup
  // local var
  float DxTxx,DxTyy,DxTzz,DxTxy,DxTxz,DxTyz,DxVx,DxVy,DxVz;
  float DyTxx,DyTyy,DyTzz,DyTxy,DyTxz,DyTyz,DyVx,DyVy,DyVz;
  float DzTxx,DzTyy,DzTzz,DzTxy,DzTxz,DzTyz,DzVx,DzVy,DzVz;
  float slw;
  float c11,c12,c13,c14,c15,c16;
  float     c22,c23,c24,c25,c26;
  float         c33,c34,c35,c36;
  float             c44,c45,c46;
  float                 c55,c56;
  float                     c66;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;

  float * Vx_ptr;
  float * Vy_ptr;
  float * Vz_ptr;
  float * Txx_ptr;
  float * Txy_ptr;
  float * Txz_ptr;
  float * Tyy_ptr;
  float * Tzz_ptr;
  float * Tyz_ptr;


  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  size_t iz = blockIdx.z * blockDim.z + threadIdx.z;

  // caclu all points
  if(ix<ni && iy<nj && iz<nk)
  {
    size_t iptr = (ix+ni1) + (iy+nj1) * siz_line + (iz+nk1) * siz_slice;

    Vx_ptr = Vx + iptr;
    Vy_ptr = Vy + iptr;
    Vz_ptr = Vz + iptr;
    Txx_ptr = Txx + iptr;
    Tyy_ptr = Tyy + iptr;
    Tzz_ptr = Tzz + iptr;
    Txz_ptr = Txz + iptr;
    Tyz_ptr = Tyz + iptr;
    Txy_ptr = Txy + iptr;

    // Vx derivatives
    M_FD_SHIFT_PTR_MACDRP(DxVx, Vx_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyVx, Vx_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzVx, Vx_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Vy derivatives
    M_FD_SHIFT_PTR_MACDRP(DxVy, Vy_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyVy, Vy_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzVy, Vy_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Vz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxVz, Vz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyVz, Vz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzVz, Vz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Txx derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTxx, Txx_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTxx, Txx_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTxx, Txx_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Tyy derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTyy, Tyy_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTyy, Tyy_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTyy, Tyy_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Tzz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTzz, Tzz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTzz, Tzz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTzz, Tzz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Txz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTxz, Txz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTxz, Txz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTxz, Txz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Tyz derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTyz, Tyz_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTyz, Tyz_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTyz, Tyz_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // Txy derivatives
    M_FD_SHIFT_PTR_MACDRP(DxTxy, Txy_ptr, fdx_len, lfdx_shift, lfdx_coef);
    M_FD_SHIFT_PTR_MACDRP(DyTxy, Txy_ptr, fdy_len, lfdy_shift, lfdy_coef);
    M_FD_SHIFT_PTR_MACDRP(DzTxy, Txy_ptr, fdz_len, lfdz_shift, lfdz_coef);

    // metric
    xix = xi_x[iptr];
    xiy = xi_y[iptr];
    xiz = xi_z[iptr];
    etx = et_x[iptr];
    ety = et_y[iptr];
    etz = et_z[iptr];
    ztx = zt_x[iptr];
    zty = zt_y[iptr];
    ztz = zt_z[iptr];

    // medium
    slw = slw3d[iptr];
    c11 = c11d[iptr];
    c12 = c12d[iptr];
    c13 = c13d[iptr];
    c14 = c14d[iptr];
    c15 = c15d[iptr];
    c16 = c16d[iptr];
    c22 = c22d[iptr];
    c23 = c23d[iptr];
    c24 = c24d[iptr];
    c25 = c25d[iptr];
    c26 = c26d[iptr];
    c33 = c33d[iptr];
    c34 = c34d[iptr];
    c35 = c35d[iptr];
    c36 = c36d[iptr];
    c44 = c44d[iptr];
    c45 = c45d[iptr];
    c46 = c46d[iptr];
    c55 = c55d[iptr];
    c56 = c56d[iptr];
    c66 = c66d[iptr];

    // moment equation
    hVx[iptr] = slw*( xix*DxTxx + xiy*DxTxy + xiz*DxTxz  
                     +etx*DyTxx + ety*DyTxy + etz*DyTxz 
                     +ztx*DzTxx + zty*DzTxy + ztz*DzTxz );
    hVy[iptr] = slw*( xix*DxTxy + xiy*DxTyy + xiz*DxTyz
                     +etx*DyTxy + ety*DyTyy + etz*DyTyz
                     +ztx*DzTxy + zty*DzTyy + ztz*DzTyz );
    hVz[iptr] = slw*( xix*DxTxz + xiy*DxTyz + xiz*DxTzz 
                     +etx*DyTxz + ety*DyTyz + etz*DyTzz
                     +ztx*DzTxz + zty*DzTyz + ztz*DzTzz );

    // Hooke's equatoin

	  hTxx[iptr] = (c11*xix + c16*xiy + c15*xiz) * DxVx + (c16*xix + c12*xiy + c14*xiz) * DxVy + (c15*xix + c14*xiy + c13*xiz) * DxVz
               + (c11*etx + c16*ety + c15*etz) * DyVx + (c16*etx + c12*ety + c14*etz) * DyVy + (c15*etx + c14*ety + c13*etz) * DyVz
               + (c11*ztx + c16*zty + c15*ztz) * DzVx + (c16*ztx + c12*zty + c14*ztz) * DzVy + (c15*ztx + c14*zty + c13*ztz) * DzVz;
    
    hTyy[iptr] = (c12*xix + c26*xiy + c25*xiz) * DxVx + (c26*xix + c22*xiy + c24*xiz) * DxVy + (c25*xix + c24*xiy + c23*xiz) * DxVz
               + (c12*etx + c26*ety + c25*etz) * DyVx + (c26*etx + c22*ety + c24*etz) * DyVy + (c25*etx + c24*ety + c23*etz) * DyVz
               + (c12*ztx + c26*zty + c25*ztz) * DzVx + (c26*ztx + c22*zty + c24*ztz) * DzVy + (c25*ztx + c24*zty + c23*ztz) * DzVz;
    
    hTzz[iptr] = (c13*xix + c36*xiy + c35*xiz) * DxVx + (c36*xix + c23*xiy + c34*xiz) * DxVy + (c35*xix + c34*xiy + c33*xiz) * DxVz
               + (c13*etx + c36*ety + c35*etz) * DyVx + (c36*etx + c23*ety + c34*etz) * DyVy + (c35*etx + c34*ety + c33*etz) * DyVz
               + (c13*ztx + c36*zty + c35*ztz) * DzVx + (c36*ztx + c23*zty + c34*ztz) * DzVy + (c35*ztx + c34*zty + c33*ztz) * DzVz;
  

    hTyz[iptr] = (c14*xix + c46*xiy + c45*xiz) * DxVx + (c46*xix + c24*xiy + c44*xiz) * DxVy + (c45*xix + c44*xiy + c34*xiz) * DxVz
               + (c14*etx + c46*ety + c45*etz) * DyVx + (c46*etx + c24*ety + c44*etz) * DyVy + (c45*etx + c44*ety + c34*etz) * DyVz
               + (c14*ztx + c46*zty + c45*ztz) * DzVx + (c46*ztx + c24*zty + c44*ztz) * DzVy + (c45*ztx + c44*zty + c34*ztz) * DzVz;
  
    hTxz[iptr] = (c15*xix + c56*xiy + c55*xiz) * DxVx + (c56*xix + c25*xiy + c45*xiz) * DxVy + (c55*xix + c45*xiy + c35*xiz) * DxVz
               + (c15*etx + c56*ety + c55*etz) * DyVx + (c56*etx + c25*ety + c45*etz) * DyVy + (c55*etx + c45*ety + c35*etz) * DyVz
               + (c15*ztx + c56*zty + c55*ztz) * DzVx + (c56*ztx + c25*zty + c45*ztz) * DzVy + (c55*ztx + c45*zty + c35*ztz) * DzVz;
  
    hTxy[iptr] = (c16*xix + c66*xiy + c56*xiz) * DxVx + (c66*xix + c26*xiy + c46*xiz) * DxVy + (c56*xix + c46*xiy + c36*xiz) * DxVz
               + (c16*etx + c66*ety + c56*etz) * DyVx + (c66*etx + c26*ety + c46*etz) * DyVy + (c56*etx + c46*ety + c36*etz) * DyVz
               + (c16*ztx + c66*zty + c56*ztz) * DzVx + (c66*ztx + c26*zty + c46*ztz) * DzVy + (c56*ztx + c46*zty + c36*ztz) * DzVz;
  }

  return;
}

/*******************************************************************************
 * free surface boundary
 ******************************************************************************/

/*
 * implement vlow boundary
 */

__global__ void
sv_eq1st_curv_col_el_aniso_rhs_vlow_z2_gpu(
    float *  Vx , float *  Vy , float *  Vz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float * c11d, float * c12d, float * c13d,
    float * c14d, float * c15d, float * c16d,
                  float * c22d, float * c23d,
    float * c24d, float * c25d, float * c26d,
                                float * c33d,
    float * c34d, float * c35d, float * c36d,
    float * c44d, float * c45d, float * c46d,
                  float * c55d, float * c56d,
                                float * c66d,
                                float * slw3d,
    float * matVx2Vz, float * matVy2Vz,
    int ni1, int ni, int nj1, int nj, int nk1, int nk2,
    size_t siz_line, size_t siz_slice,
    int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
    int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
    int num_of_fdz_op, int fdz_max_len, int * fdz_len,
    float *lfdz_coef_all, size_t *lfdz_shift_all,
    const int myid, const int verbose)
{
  // local var
  int k;
  int n_fd; // loop var for fd
  int lfdz_len;
  // local var
  float DxVx,DxVy,DxVz;
  float DyVx,DyVy,DyVz;
  float DzVx,DzVy,DzVz;
  float slw;
  float c11,c12,c13,c14,c15,c16;
  float     c22,c23,c24,c25,c26;
  float         c33,c34,c35,c36;
  float             c44,c45,c46;
  float                 c55,c56;
  float                     c66;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;

  float lfdz_coef[5] = {0.0};
  int   lfdz_shift[5] = {0};
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;

  // loop near surface layers
  for (int n=0; n < num_of_fdz_op-1; n++)
  {
    // conver to k index, from surface to inner
    k = nk2 - n;
    // get pos and len for this point
    lfdz_len  = fdz_len[n];
    for (n_fd = 0; n_fd < lfdz_len ; n_fd++) {
      lfdz_shift[n_fd] = lfdz_shift_all[n*fdz_max_len+n_fd];
      lfdz_coef [n_fd]  = lfdz_coef_all [n*fdz_max_len+n_fd];
    }
    if(ix<ni && iy<nj)
    {
      size_t iptr   = (ix+ni1) + (iy+nj1) * siz_line + k * siz_slice;

      // metric
      xix = xi_x[iptr];
      xiy = xi_y[iptr];
      xiz = xi_z[iptr];
      etx = et_x[iptr];
      ety = et_y[iptr];
      etz = et_z[iptr];
      ztx = zt_x[iptr];
      zty = zt_y[iptr];
      ztz = zt_z[iptr];

      // medium
      slw = slw3d[iptr];
      c11 = c11d[iptr];
      c12 = c12d[iptr];
      c13 = c13d[iptr];
      c14 = c14d[iptr];
      c15 = c15d[iptr];
      c16 = c16d[iptr];
      c22 = c22d[iptr];
      c23 = c23d[iptr];
      c24 = c24d[iptr];
      c25 = c25d[iptr];
      c26 = c26d[iptr];
      c33 = c33d[iptr];
      c34 = c34d[iptr];
      c35 = c35d[iptr];
      c36 = c36d[iptr];
      c44 = c44d[iptr];
      c45 = c45d[iptr];
      c46 = c46d[iptr];
      c55 = c55d[iptr];
      c56 = c56d[iptr];
      c66 = c66d[iptr];

      // Vx derivatives
      M_FD_SHIFT(DxVx, Vx, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DyVx, Vx, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      // Vy derivatives
      M_FD_SHIFT(DxVy, Vy, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DyVy, Vy, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      // Vz derivatives
      M_FD_SHIFT(DxVz, Vz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DyVz, Vz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      if (k==nk2) // at surface, convert
      {
        size_t ij = ((ix+ni1) + (iy+nj1) * siz_line)*9;
        DzVx = matVx2Vz[ij+3*0+0] * DxVx
             + matVx2Vz[ij+3*0+1] * DxVy
             + matVx2Vz[ij+3*0+2] * DxVz
             + matVy2Vz[ij+3*0+0] * DyVx
             + matVy2Vz[ij+3*0+1] * DyVy
             + matVy2Vz[ij+3*0+2] * DyVz;

        DzVy = matVx2Vz[ij+3*1+0] * DxVx
             + matVx2Vz[ij+3*1+1] * DxVy
             + matVx2Vz[ij+3*1+2] * DxVz
             + matVy2Vz[ij+3*1+0] * DyVx
             + matVy2Vz[ij+3*1+1] * DyVy
             + matVy2Vz[ij+3*1+2] * DyVz;

        DzVz = matVx2Vz[ij+3*2+0] * DxVx
             + matVx2Vz[ij+3*2+1] * DxVy
             + matVx2Vz[ij+3*2+2] * DxVz
             + matVy2Vz[ij+3*2+0] * DyVx
             + matVy2Vz[ij+3*2+1] * DyVy
             + matVy2Vz[ij+3*2+2] * DyVz;
      }
      else // lower than surface, lower order
      {
        M_FD_SHIFT(DzVx, Vx, iptr, lfdz_len, lfdz_shift, lfdz_coef, n_fd);
        M_FD_SHIFT(DzVy, Vy, iptr, lfdz_len, lfdz_shift, lfdz_coef, n_fd);
        M_FD_SHIFT(DzVz, Vz, iptr, lfdz_len, lfdz_shift, lfdz_coef, n_fd);
      }

      // Hooke's equatoin
	    hTxx[iptr] = (c11*xix + c16*xiy + c15*xiz) * DxVx + (c16*xix + c12*xiy + c14*xiz) * DxVy + (c15*xix + c14*xiy + c13*xiz) * DxVz
                 + (c11*etx + c16*ety + c15*etz) * DyVx + (c16*etx + c12*ety + c14*etz) * DyVy + (c15*etx + c14*ety + c13*etz) * DyVz
                 + (c11*ztx + c16*zty + c15*ztz) * DzVx + (c16*ztx + c12*zty + c14*ztz) * DzVy + (c15*ztx + c14*zty + c13*ztz) * DzVz;
      
      hTyy[iptr] = (c12*xix + c26*xiy + c25*xiz) * DxVx + (c26*xix + c22*xiy + c24*xiz) * DxVy + (c25*xix + c24*xiy + c23*xiz) * DxVz
                 + (c12*etx + c26*ety + c25*etz) * DyVx + (c26*etx + c22*ety + c24*etz) * DyVy + (c25*etx + c24*ety + c23*etz) * DyVz
                 + (c12*ztx + c26*zty + c25*ztz) * DzVx + (c26*ztx + c22*zty + c24*ztz) * DzVy + (c25*ztx + c24*zty + c23*ztz) * DzVz;
     
      hTzz[iptr] = (c13*xix + c36*xiy + c35*xiz) * DxVx + (c36*xix + c23*xiy + c34*xiz) * DxVy + (c35*xix + c34*xiy + c33*xiz) * DxVz
                 + (c13*etx + c36*ety + c35*etz) * DyVx + (c36*etx + c23*ety + c34*etz) * DyVy + (c35*etx + c34*ety + c33*etz) * DyVz
                 + (c13*ztx + c36*zty + c35*ztz) * DzVx + (c36*ztx + c23*zty + c34*ztz) * DzVy + (c35*ztx + c34*zty + c33*ztz) * DzVz;
  

      hTyz[iptr] = (c14*xix + c46*xiy + c45*xiz) * DxVx + (c46*xix + c24*xiy + c44*xiz) * DxVy + (c45*xix + c44*xiy + c34*xiz) * DxVz
                 + (c14*etx + c46*ety + c45*etz) * DyVx + (c46*etx + c24*ety + c44*etz) * DyVy + (c45*etx + c44*ety + c34*etz) * DyVz
                 + (c14*ztx + c46*zty + c45*ztz) * DzVx + (c46*ztx + c24*zty + c44*ztz) * DzVy + (c45*ztx + c44*zty + c34*ztz) * DzVz;
  
      hTxz[iptr] = (c15*xix + c56*xiy + c55*xiz) * DxVx + (c56*xix + c25*xiy + c45*xiz) * DxVy + (c55*xix + c45*xiy + c35*xiz) * DxVz
                 + (c15*etx + c56*ety + c55*etz) * DyVx + (c56*etx + c25*ety + c45*etz) * DyVy + (c55*etx + c45*ety + c35*etz) * DyVz
                 + (c15*ztx + c56*zty + c55*ztz) * DzVx + (c56*ztx + c25*zty + c45*ztz) * DzVy + (c55*ztx + c45*zty + c35*ztz) * DzVz;
  
      hTxy[iptr] = (c16*xix + c66*xiy + c56*xiz) * DxVx + (c66*xix + c26*xiy + c46*xiz) * DxVy + (c56*xix + c46*xiy + c36*xiz) * DxVz
                 + (c16*etx + c66*ety + c56*etz) * DyVx + (c66*etx + c26*ety + c46*etz) * DyVy + (c56*etx + c46*ety + c36*etz) * DyVz
                 + (c16*ztx + c66*zty + c56*ztz) * DzVx + (c66*ztx + c26*zty + c46*ztz) * DzVy + (c56*ztx + c46*zty + c36*ztz) * DzVz;
    }
  }

  return;
}

/*******************************************************************************
 * CFS-PML boundary
 ******************************************************************************/

/*
 * cfspml, reference to each pml var inside function
 */

void
sv_eq1st_curv_col_el_aniso_rhs_cfspml(
    float *  Vx , float *  Vy , float *  Vz ,
    float *  Txx, float *  Tyy, float *  Tzz,
    float *  Txz, float *  Tyz, float *  Txy,
    float * hVx , float * hVy , float * hVz ,
    float * hTxx, float * hTyy, float * hTzz,
    float * hTxz, float * hTyz, float * hTxy,
    float * xi_x, float * xi_y, float * xi_z,
    float * et_x, float * et_y, float * et_z,
    float * zt_x, float * zt_y, float * zt_z,
    float * c11d, float * c12d, float * c13d,
    float * c14d, float * c15d, float * c16d,
                  float * c22d, float * c23d,
    float * c24d, float * c25d, float * c26d,
                                float * c33d,
    float * c34d, float * c35d, float * c36d,
    float * c44d, float * c45d, float * c46d,
                  float * c55d, float * c56d,
                                float * c66d,
                                float * slw3d,
    int nk2, size_t siz_line, size_t siz_slice,
    int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
    int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
    int fdz_len, size_t * lfdz_shift, float * lfdz_coef,
    bdrypml_t bdrypml, bdryfree_t bdryfree,
    const int myid, const int verbose)
{
  // check each side
  for (int idim=0; idim<CONST_NDIM; idim++)
  {
    for (int iside=0; iside<2; iside++)
    {
      // skip to next face if not cfspml
      if (bdrypml.is_at_sides[idim][iside] == 0) continue;

      // get index into local var
      int abs_ni1 = bdrypml.ni1[idim][iside];
      int abs_ni2 = bdrypml.ni2[idim][iside];
      int abs_nj1 = bdrypml.nj1[idim][iside];
      int abs_nj2 = bdrypml.nj2[idim][iside];
      int abs_nk1 = bdrypml.nk1[idim][iside];
      int abs_nk2 = bdrypml.nk2[idim][iside];

      
      int abs_ni = abs_ni2-abs_ni1+1; 
      int abs_nj = abs_nj2-abs_nj1+1; 
      int abs_nk = abs_nk2-abs_nk1+1; 
      {
        dim3 block(8,4,4);
        dim3 grid;
        grid.x = (abs_ni+block.x-1)/block.x;
        grid.y = (abs_nj+block.y-1)/block.y;
        grid.z = (abs_nk+block.z-1)/block.z;

        sv_eq1st_curv_col_el_aniso_rhs_cfspml_gpu <<<grid, block>>> (
                                idim, iside, Vx , Vy , Vz , 
                                Txx, Tyy, Tzz, Txz, Tyz, Txy, 
                                hVx, hVy, hVz, hTxx, hTyy, hTzz, 
                                hTxz, hTyz, hTxy, xi_x, xi_y, xi_z,
                                et_x, et_y, et_z, zt_x, zt_y, zt_z, 
                                c11d,c12d,c13d,c14d,c15d,c16d,
                                     c22d,c23d,c24d,c25d,c26d,
                                          c33d,c34d,c35d,c36d,
                                               c44d,c45d,c46d,
                                                    c55d,c56d,
                                                    c66d,slw3d,
                                nk2, siz_line, siz_slice,
                                fdx_len, lfdx_shift,  lfdx_coef,
                                fdy_len, lfdy_shift,  lfdy_coef,
                                fdz_len, lfdz_shift,  lfdz_coef,
                                bdrypml, bdryfree, myid, verbose);
        //hipDeviceSynchronize();
      }
    } // iside
  } // idim

  return;
}


__global__ void
sv_eq1st_curv_col_el_aniso_rhs_cfspml_gpu(int idim, int iside,
                                        float *  Vx , float *  Vy , float *  Vz ,
                                        float *  Txx, float *  Tyy, float *  Tzz,
                                        float *  Txz, float *  Tyz, float *  Txy,
                                        float * hVx , float * hVy , float * hVz ,
                                        float * hTxx, float * hTyy, float * hTzz,
                                        float * hTxz, float * hTyz, float * hTxy,
                                        float * xi_x, float * xi_y, float * xi_z,
                                        float * et_x, float * et_y, float * et_z,
                                        float * zt_x, float * zt_y, float * zt_z,
                                        float * c11d, float * c12d, float * c13d,
                                        float * c14d, float * c15d, float * c16d,
                                                      float * c22d, float * c23d,
                                        float * c24d, float * c25d, float * c26d,
                                                                    float * c33d,
                                        float * c34d, float * c35d, float * c36d,
                                        float * c44d, float * c45d, float * c46d,
                                                      float * c55d, float * c56d,
                                                                    float * c66d,
                                                                    float * slw3d,
                                        int nk2, size_t siz_line, size_t siz_slice,
                                        int fdx_len, size_t * lfdx_shift, float * lfdx_coef,
                                        int fdy_len, size_t * lfdy_shift, float * lfdy_coef,
                                        int fdz_len, size_t * lfdz_shift, float * lfdz_coef,
                                        bdrypml_t bdrypml, bdryfree_t bdryfree,
                                        const int myid, const int verbose)
{
  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  size_t iz = blockIdx.z * blockDim.z + threadIdx.z;
  float *matVx2Vz = bdryfree.matVx2Vz2;
  float *matVy2Vz = bdryfree.matVy2Vz2;
  // local
  size_t iptr, iptr_a;
  float coef_A, coef_B, coef_D, coef_B_minus_1;
  // loop var for fd
  int n_fd;

  // get index into local var
  int abs_ni1 = bdrypml.ni1[idim][iside];
  int abs_ni2 = bdrypml.ni2[idim][iside];
  int abs_nj1 = bdrypml.nj1[idim][iside];
  int abs_nj2 = bdrypml.nj2[idim][iside];
  int abs_nk1 = bdrypml.nk1[idim][iside];
  int abs_nk2 = bdrypml.nk2[idim][iside];

  
  int abs_ni = abs_ni2-abs_ni1+1; 
  int abs_nj = abs_nj2-abs_nj1+1; 
  int abs_nk = abs_nk2-abs_nk1+1; 

  // val on point
  float DxTxx,DxTyy,DxTzz,DxTxy,DxTxz,DxTyz,DxVx,DxVy,DxVz;
  float DyTxx,DyTyy,DyTzz,DyTxy,DyTxz,DyTyz,DyVx,DyVy,DyVz;
  float DzTxx,DzTyy,DzTzz,DzTxy,DzTxz,DzTyz,DzVx,DzVy,DzVz;
  float slw;
  float c11,c12,c13,c14,c15,c16;
  float     c22,c23,c24,c25,c26;
  float         c33,c34,c35,c36;
  float             c44,c45,c46;
  float                 c55,c56;
  float                     c66;
  float xix,xiy,xiz,etx,ety,etz,ztx,zty,ztz;
  float hVx_rhs,hVy_rhs,hVz_rhs;
  float hTxx_rhs,hTyy_rhs,hTzz_rhs,hTxz_rhs,hTyz_rhs,hTxy_rhs;
  // for free surface
  float Dx_DzVx,Dy_DzVx,Dx_DzVy,Dy_DzVy,Dx_DzVz,Dy_DzVz;
  // get coef for this face
  float * ptr_coef_A = bdrypml.A[idim][iside];
  float * ptr_coef_B = bdrypml.B[idim][iside];
  float * ptr_coef_D = bdrypml.D[idim][iside];

  bdrypml_auxvar_t *auxvar = &(bdrypml.auxvar[idim][iside]);

  // get pml vars
  float * abs_vars_cur = auxvar->cur;
  float * abs_vars_rhs = auxvar->rhs;

  float * pml_Vx   = abs_vars_cur + auxvar->Vx_pos;
  float * pml_Vy   = abs_vars_cur + auxvar->Vy_pos;
  float * pml_Vz   = abs_vars_cur + auxvar->Vz_pos;
  float * pml_Txx  = abs_vars_cur + auxvar->Txx_pos;
  float * pml_Tyy  = abs_vars_cur + auxvar->Tyy_pos;
  float * pml_Tzz  = abs_vars_cur + auxvar->Tzz_pos;
  float * pml_Txz  = abs_vars_cur + auxvar->Txz_pos;
  float * pml_Tyz  = abs_vars_cur + auxvar->Tyz_pos;
  float * pml_Txy  = abs_vars_cur + auxvar->Txy_pos;

  float * pml_hVx  = abs_vars_rhs + auxvar->Vx_pos;
  float * pml_hVy  = abs_vars_rhs + auxvar->Vy_pos;
  float * pml_hVz  = abs_vars_rhs + auxvar->Vz_pos;
  float * pml_hTxx = abs_vars_rhs + auxvar->Txx_pos;
  float * pml_hTyy = abs_vars_rhs + auxvar->Tyy_pos;
  float * pml_hTzz = abs_vars_rhs + auxvar->Tzz_pos;
  float * pml_hTxz = abs_vars_rhs + auxvar->Txz_pos;
  float * pml_hTyz = abs_vars_rhs + auxvar->Tyz_pos;
  float * pml_hTxy = abs_vars_rhs + auxvar->Txy_pos;


  // for each dim
  if (idim == 0 ) // x direction
  {
    if(ix<abs_ni  && iy<abs_nj && iz<abs_nk)
    {
      iptr_a = iz*(abs_nj*abs_ni) + iy*abs_ni + ix;
      iptr   = (ix + abs_ni1) + (iy+abs_nj1) * siz_line + (iz+abs_nk1) * siz_slice;
      // pml coefs
      // int abs_i = ix;
      coef_D = ptr_coef_D[ix];
      coef_A = ptr_coef_A[ix];
      coef_B = ptr_coef_B[ix];
      coef_B_minus_1 = coef_B - 1.0;

      // metric
      xix = xi_x[iptr];
      xiy = xi_y[iptr];
      xiz = xi_z[iptr];

      // medium
      slw = slw3d[iptr];
      c11 = c11d[iptr];
      c12 = c12d[iptr];
      c13 = c13d[iptr];
      c14 = c14d[iptr];
      c15 = c15d[iptr];
      c16 = c16d[iptr];
      c22 = c22d[iptr];
      c23 = c23d[iptr];
      c24 = c24d[iptr];
      c25 = c25d[iptr];
      c26 = c26d[iptr];
      c33 = c33d[iptr];
      c34 = c34d[iptr];
      c35 = c35d[iptr];
      c36 = c36d[iptr];
      c44 = c44d[iptr];
      c45 = c45d[iptr];
      c46 = c46d[iptr];
      c55 = c55d[iptr];
      c56 = c56d[iptr];
      c66 = c66d[iptr];

      // xi derivatives
      M_FD_SHIFT(DxVx , Vx , iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxVy , Vy , iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxVz , Vz , iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTxx, Txx, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTyy, Tyy, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTzz, Tzz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTxz, Txz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTyz, Tyz, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);
      M_FD_SHIFT(DxTxy, Txy, iptr, fdx_len, lfdx_shift, lfdx_coef, n_fd);

      // combine for corr and aux vars
       hVx_rhs = slw * ( xix*DxTxx + xiy*DxTxy + xiz*DxTxz );
       hVy_rhs = slw * ( xix*DxTxy + xiy*DxTyy + xiz*DxTyz );
       hVz_rhs = slw * ( xix*DxTxz + xiy*DxTyz + xiz*DxTzz );
      hTxx_rhs = (c11*xix+c16*xiy+c15*xiz)*DxVx + (c16*xix+c12*xiy+c14*xiz)*DxVy + (c15*xix+c14*xiy+c13*xiz)*DxVz; 
      hTyy_rhs = (c12*xix+c26*xiy+c25*xiz)*DxVx + (c26*xix+c22*xiy+c24*xiz)*DxVy + (c25*xix+c24*xiy+c23*xiz)*DxVz;
      hTzz_rhs = (c13*xix+c36*xiy+c35*xiz)*DxVx + (c36*xix+c23*xiy+c34*xiz)*DxVy + (c35*xix+c34*xiy+c33*xiz)*DxVz;
      hTyz_rhs = (c14*xix+c46*xiy+c45*xiz)*DxVx + (c46*xix+c24*xiy+c44*xiz)*DxVy + (c45*xix+c44*xiy+c34*xiz)*DxVz;
      hTxz_rhs = (c15*xix+c56*xiy+c55*xiz)*DxVx + (c56*xix+c25*xiy+c45*xiz)*DxVy + (c55*xix+c45*xiy+c35*xiz)*DxVz;
      hTxy_rhs = (c16*xix+c66*xiy+c56*xiz)*DxVx + (c66*xix+c26*xiy+c46*xiz)*DxVy + (c56*xix+c46*xiy+c36*xiz)*DxVz;

      // 1: make corr to moment equation
      hVx[iptr] += coef_B_minus_1 * hVx_rhs - coef_B * pml_Vx[iptr_a];
      hVy[iptr] += coef_B_minus_1 * hVy_rhs - coef_B * pml_Vy[iptr_a];
      hVz[iptr] += coef_B_minus_1 * hVz_rhs - coef_B * pml_Vz[iptr_a];

      // make corr to Hooke's equatoin
      hTxx[iptr] += coef_B_minus_1 * hTxx_rhs - coef_B * pml_Txx[iptr_a];
      hTyy[iptr] += coef_B_minus_1 * hTyy_rhs - coef_B * pml_Tyy[iptr_a];
      hTzz[iptr] += coef_B_minus_1 * hTzz_rhs - coef_B * pml_Tzz[iptr_a];
      hTxz[iptr] += coef_B_minus_1 * hTxz_rhs - coef_B * pml_Txz[iptr_a];
      hTyz[iptr] += coef_B_minus_1 * hTyz_rhs - coef_B * pml_Tyz[iptr_a];
      hTxy[iptr] += coef_B_minus_1 * hTxy_rhs - coef_B * pml_Txy[iptr_a];
      
      // 2: aux var
      //   a1 = alpha + d / beta, dealt in abs_set_cfspml
      pml_hVx[iptr_a]  = coef_D * hVx_rhs  - coef_A * pml_Vx[iptr_a];
      pml_hVy[iptr_a]  = coef_D * hVy_rhs  - coef_A * pml_Vy[iptr_a];
      pml_hVz[iptr_a]  = coef_D * hVz_rhs  - coef_A * pml_Vz[iptr_a];
      pml_hTxx[iptr_a] = coef_D * hTxx_rhs - coef_A * pml_Txx[iptr_a];
      pml_hTyy[iptr_a] = coef_D * hTyy_rhs - coef_A * pml_Tyy[iptr_a];
      pml_hTzz[iptr_a] = coef_D * hTzz_rhs - coef_A * pml_Tzz[iptr_a];
      pml_hTxz[iptr_a] = coef_D * hTxz_rhs - coef_A * pml_Txz[iptr_a];
      pml_hTyz[iptr_a] = coef_D * hTyz_rhs - coef_A * pml_Tyz[iptr_a];
      pml_hTxy[iptr_a] = coef_D * hTxy_rhs - coef_A * pml_Txy[iptr_a];

      // add contributions from free surface condition
      //  not consider timg because conflict with main cfspml,
      //     need to revise in the future if required
      if (bdryfree.is_at_sides[CONST_NDIM-1][1]==1 && (iz+abs_nk1)==nk2)
      {
        // zeta derivatives
        size_t ij = ((ix+abs_ni1) + (iy+abs_nj1) * siz_line)*9;
        Dx_DzVx = matVx2Vz[ij+3*0+0] * DxVx
                + matVx2Vz[ij+3*0+1] * DxVy
                + matVx2Vz[ij+3*0+2] * DxVz;

        Dx_DzVy = matVx2Vz[ij+3*1+0] * DxVx
                + matVx2Vz[ij+3*1+1] * DxVy
                + matVx2Vz[ij+3*1+2] * DxVz;

        Dx_DzVz = matVx2Vz[ij+3*2+0] * DxVx
                + matVx2Vz[ij+3*2+1] * DxVy
                + matVx2Vz[ij+3*2+2] * DxVz;

        // metric
        ztx = zt_x[iptr];
        zty = zt_y[iptr];
        ztz = zt_z[iptr];

        // keep xi derivative terms, including free surface convered
        hTxx_rhs = (c11*ztx+c16*zty+c15*ztz)*Dx_DzVx + (c16*ztx+c12*zty+c14*ztz)*Dx_DzVy + (c15*ztx+c14*zty+c13*ztz)*Dx_DzVz; 
        hTyy_rhs = (c12*ztx+c26*zty+c25*ztz)*Dx_DzVx + (c26*ztx+c22*zty+c24*ztz)*Dx_DzVy + (c25*ztx+c24*zty+c23*ztz)*Dx_DzVz;
        hTzz_rhs = (c13*ztx+c36*zty+c35*ztz)*Dx_DzVx + (c36*ztx+c23*zty+c34*ztz)*Dx_DzVy + (c35*ztx+c34*zty+c33*ztz)*Dx_DzVz;
        hTyz_rhs = (c14*ztx+c46*zty+c45*ztz)*Dx_DzVx + (c46*ztx+c24*zty+c44*ztz)*Dx_DzVy + (c45*ztx+c44*zty+c34*ztz)*Dx_DzVz;
        hTxz_rhs = (c15*ztx+c56*zty+c55*ztz)*Dx_DzVx + (c56*ztx+c25*zty+c45*ztz)*Dx_DzVy + (c55*ztx+c45*zty+c35*ztz)*Dx_DzVz;
        hTxy_rhs = (c16*ztx+c66*zty+c56*ztz)*Dx_DzVx + (c66*ztx+c26*zty+c46*ztz)*Dx_DzVy + (c56*ztx+c46*zty+c36*ztz)*Dx_DzVz;

        // make corr to Hooke's equatoin
        hTxx[iptr] += (coef_B - 1.0) * hTxx_rhs;
        hTyy[iptr] += (coef_B - 1.0) * hTyy_rhs;
        hTzz[iptr] += (coef_B - 1.0) * hTzz_rhs;
        hTxz[iptr] += (coef_B - 1.0) * hTxz_rhs;
        hTyz[iptr] += (coef_B - 1.0) * hTyz_rhs;
        hTxy[iptr] += (coef_B - 1.0) * hTxy_rhs;

        // aux var
        //   a1 = alpha + d / beta, dealt in abs_set_cfspml
        pml_hTxx[iptr_a] += coef_D * hTxx_rhs;
        pml_hTyy[iptr_a] += coef_D * hTyy_rhs;
        pml_hTzz[iptr_a] += coef_D * hTzz_rhs;
        pml_hTxz[iptr_a] += coef_D * hTxz_rhs;
        pml_hTyz[iptr_a] += coef_D * hTyz_rhs;
        pml_hTxy[iptr_a] += coef_D * hTxy_rhs;
      } // if nk2
    }
  }
  else if (idim == 1) // y direction
  {
    if(ix<abs_ni  && iy<abs_nj && iz<abs_nk)
    {
      iptr_a = iz*(abs_nj*abs_ni) + iy*abs_ni + ix;
      iptr   = (ix + abs_ni1) + (iy+abs_nj1)*siz_line + (iz+abs_nk1) * siz_slice;

      // pml coefs
      // int abs_j = iy;
      coef_D = ptr_coef_D[iy];
      coef_A = ptr_coef_A[iy];
      coef_B = ptr_coef_B[iy];
      coef_B_minus_1 = coef_B - 1.0;

      // metric
      etx = et_x[iptr];
      ety = et_y[iptr];
      etz = et_z[iptr];

      // medium
      slw = slw3d[iptr];
      c11 = c11d[iptr];
      c12 = c12d[iptr];
      c13 = c13d[iptr];
      c14 = c14d[iptr];
      c15 = c15d[iptr];
      c16 = c16d[iptr];
      c22 = c22d[iptr];
      c23 = c23d[iptr];
      c24 = c24d[iptr];
      c25 = c25d[iptr];
      c26 = c26d[iptr];
      c33 = c33d[iptr];
      c34 = c34d[iptr];
      c35 = c35d[iptr];
      c36 = c36d[iptr];
      c44 = c44d[iptr];
      c45 = c45d[iptr];
      c46 = c46d[iptr];
      c55 = c55d[iptr];
      c56 = c56d[iptr];
      c66 = c66d[iptr];

      // et derivatives
      M_FD_SHIFT(DyVx , Vx , iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyVy , Vy , iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyVz , Vz , iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTxx, Txx, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTyy, Tyy, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTzz, Tzz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTxz, Txz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTyz, Tyz, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);
      M_FD_SHIFT(DyTxy, Txy, iptr, fdy_len, lfdy_shift, lfdy_coef, n_fd);

      // combine for corr and aux vars
       hVx_rhs = slw * ( etx*DyTxx + ety*DyTxy + etz*DyTxz );
       hVy_rhs = slw * ( etx*DyTxy + ety*DyTyy + etz*DyTyz );
       hVz_rhs = slw * ( etx*DyTxz + ety*DyTyz + etz*DyTzz );
      hTxx_rhs = (c11*etx+c16*ety+c15*etz)*DyVx + (c16*etx+c12*ety+c14*etz)*DyVy + (c15*etx+c14*ety+c13*etz)*DyVz; 
      hTyy_rhs = (c12*etx+c26*ety+c25*etz)*DyVx + (c26*etx+c22*ety+c24*etz)*DyVy + (c25*etx+c24*ety+c23*etz)*DyVz;
      hTzz_rhs = (c13*etx+c36*ety+c35*etz)*DyVx + (c36*etx+c23*ety+c34*etz)*DyVy + (c35*etx+c34*ety+c33*etz)*DyVz;
      hTyz_rhs = (c14*etx+c46*ety+c45*etz)*DyVx + (c46*etx+c24*ety+c44*etz)*DyVy + (c45*etx+c44*ety+c34*etz)*DyVz;
      hTxz_rhs = (c15*etx+c56*ety+c55*etz)*DyVx + (c56*etx+c25*ety+c45*etz)*DyVy + (c55*etx+c45*ety+c35*etz)*DyVz;
      hTxy_rhs = (c16*etx+c66*ety+c56*etz)*DyVx + (c66*etx+c26*ety+c46*etz)*DyVy + (c56*etx+c46*ety+c36*etz)*DyVz;

      // 1: make corr to moment equation
      hVx[iptr] += coef_B_minus_1 * hVx_rhs - coef_B * pml_Vx[iptr_a];
      hVy[iptr] += coef_B_minus_1 * hVy_rhs - coef_B * pml_Vy[iptr_a];
      hVz[iptr] += coef_B_minus_1 * hVz_rhs - coef_B * pml_Vz[iptr_a];

      // make corr to Hooke's equatoin
      hTxx[iptr] += coef_B_minus_1 * hTxx_rhs - coef_B * pml_Txx[iptr_a];
      hTyy[iptr] += coef_B_minus_1 * hTyy_rhs - coef_B * pml_Tyy[iptr_a];
      hTzz[iptr] += coef_B_minus_1 * hTzz_rhs - coef_B * pml_Tzz[iptr_a];
      hTxz[iptr] += coef_B_minus_1 * hTxz_rhs - coef_B * pml_Txz[iptr_a];
      hTyz[iptr] += coef_B_minus_1 * hTyz_rhs - coef_B * pml_Tyz[iptr_a];
      hTxy[iptr] += coef_B_minus_1 * hTxy_rhs - coef_B * pml_Txy[iptr_a];
      
      // 2: aux var
      //   a1 = alpha + d / beta, dealt in abs_set_cfspml
      pml_hVx[iptr_a]  = coef_D * hVx_rhs  - coef_A * pml_Vx[iptr_a];
      pml_hVy[iptr_a]  = coef_D * hVy_rhs  - coef_A * pml_Vy[iptr_a];
      pml_hVz[iptr_a]  = coef_D * hVz_rhs  - coef_A * pml_Vz[iptr_a];
      pml_hTxx[iptr_a] = coef_D * hTxx_rhs - coef_A * pml_Txx[iptr_a];
      pml_hTyy[iptr_a] = coef_D * hTyy_rhs - coef_A * pml_Tyy[iptr_a];
      pml_hTzz[iptr_a] = coef_D * hTzz_rhs - coef_A * pml_Tzz[iptr_a];
      pml_hTxz[iptr_a] = coef_D * hTxz_rhs - coef_A * pml_Txz[iptr_a];
      pml_hTyz[iptr_a] = coef_D * hTyz_rhs - coef_A * pml_Tyz[iptr_a];
      pml_hTxy[iptr_a] = coef_D * hTxy_rhs - coef_A * pml_Txy[iptr_a];

      // add contributions from free surface condition
      if (bdryfree.is_at_sides[CONST_NDIM-1][1]==1 && (iz+abs_nk1)==nk2)
      {
        // zeta derivatives
        size_t ij = ((ix+abs_ni1) + (iy+abs_nj1) * siz_line)*9;
        Dy_DzVx = matVy2Vz[ij+3*0+0] * DyVx
                + matVy2Vz[ij+3*0+1] * DyVy
                + matVy2Vz[ij+3*0+2] * DyVz;

        Dy_DzVy = matVy2Vz[ij+3*1+0] * DyVx
                + matVy2Vz[ij+3*1+1] * DyVy
                + matVy2Vz[ij+3*1+2] * DyVz;

        Dy_DzVz = matVy2Vz[ij+3*2+0] * DyVx
                + matVy2Vz[ij+3*2+1] * DyVy
                + matVy2Vz[ij+3*2+2] * DyVz;

        // metric
        ztx = zt_x[iptr];
        zty = zt_y[iptr];
        ztz = zt_z[iptr];

        // keep eta derivative terms, including free surface convered
        hTxx_rhs = (c11*ztx+c16*zty+c15*ztz)*Dy_DzVx + (c16*ztx+c12*zty+c14*ztz)*Dy_DzVy + (c15*ztx+c14*zty+c13*ztz)*Dy_DzVz; 
        hTyy_rhs = (c12*ztx+c26*zty+c25*ztz)*Dy_DzVx + (c26*ztx+c22*zty+c24*ztz)*Dy_DzVy + (c25*ztx+c24*zty+c23*ztz)*Dy_DzVz;
        hTzz_rhs = (c13*ztx+c36*zty+c35*ztz)*Dy_DzVx + (c36*ztx+c23*zty+c34*ztz)*Dy_DzVy + (c35*ztx+c34*zty+c33*ztz)*Dy_DzVz;
        hTyz_rhs = (c14*ztx+c46*zty+c45*ztz)*Dy_DzVx + (c46*ztx+c24*zty+c44*ztz)*Dy_DzVy + (c45*ztx+c44*zty+c34*ztz)*Dy_DzVz;
        hTxz_rhs = (c15*ztx+c56*zty+c55*ztz)*Dy_DzVx + (c56*ztx+c25*zty+c45*ztz)*Dy_DzVy + (c55*ztx+c45*zty+c35*ztz)*Dy_DzVz;
        hTxy_rhs = (c16*ztx+c66*zty+c56*ztz)*Dy_DzVx + (c66*ztx+c26*zty+c46*ztz)*Dy_DzVy + (c56*ztx+c46*zty+c36*ztz)*Dy_DzVz;

        // make corr to Hooke's equatoin
        hTxx[iptr] += (coef_B - 1.0) * hTxx_rhs;
        hTyy[iptr] += (coef_B - 1.0) * hTyy_rhs;
        hTzz[iptr] += (coef_B - 1.0) * hTzz_rhs;
        hTxz[iptr] += (coef_B - 1.0) * hTxz_rhs;
        hTyz[iptr] += (coef_B - 1.0) * hTyz_rhs;
        hTxy[iptr] += (coef_B - 1.0) * hTxy_rhs;

        // aux var
        //   a1 = alpha + d / beta, dealt in abs_set_cfspml
        pml_hTxx[iptr_a] += coef_D * hTxx_rhs;
        pml_hTyy[iptr_a] += coef_D * hTyy_rhs;
        pml_hTzz[iptr_a] += coef_D * hTzz_rhs;
        pml_hTxz[iptr_a] += coef_D * hTxz_rhs;
        pml_hTyz[iptr_a] += coef_D * hTyz_rhs;
        pml_hTxy[iptr_a] += coef_D * hTxy_rhs;
      }
    }
  }
  else // z direction
  {
    if(ix<abs_ni  && iy<abs_nj && iz<abs_nk)
    {
      iptr_a = iz*(abs_nj*abs_ni) + iy*abs_ni + ix;
      iptr   = (ix + abs_ni1) + (iy+abs_nj1) * siz_line + (iz+abs_nk1) * siz_slice;
      // pml coefs
      // int abs_k = iz;
      coef_D = ptr_coef_D[iz];
      coef_A = ptr_coef_A[iz];
      coef_B = ptr_coef_B[iz];
      coef_B_minus_1 = coef_B - 1.0;

      // metric
      ztx = zt_x[iptr];
      zty = zt_y[iptr];
      ztz = zt_z[iptr];

      // medium
      slw = slw3d[iptr];
      c11 = c11d[iptr];
      c12 = c12d[iptr];
      c13 = c13d[iptr];
      c14 = c14d[iptr];
      c15 = c15d[iptr];
      c16 = c16d[iptr];
      c22 = c22d[iptr];
      c23 = c23d[iptr];
      c24 = c24d[iptr];
      c25 = c25d[iptr];
      c26 = c26d[iptr];
      c33 = c33d[iptr];
      c34 = c34d[iptr];
      c35 = c35d[iptr];
      c36 = c36d[iptr];
      c44 = c44d[iptr];
      c45 = c45d[iptr];
      c46 = c46d[iptr];
      c55 = c55d[iptr];
      c56 = c56d[iptr];
      c66 = c66d[iptr];

      // zt derivatives
      M_FD_SHIFT(DzVx , Vx , iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzVy , Vy , iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzVz , Vz , iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTxx, Txx, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTyy, Tyy, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTzz, Tzz, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTxz, Txz, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTyz, Tyz, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);
      M_FD_SHIFT(DzTxy, Txy, iptr, fdz_len, lfdz_shift, lfdz_coef, n_fd);

      // combine for corr and aux vars
       hVx_rhs = slw * ( ztx*DzTxx + zty*DzTxy + ztz*DzTxz );
       hVy_rhs = slw * ( ztx*DzTxy + zty*DzTyy + ztz*DzTyz );
       hVz_rhs = slw * ( ztx*DzTxz + zty*DzTyz + ztz*DzTzz );
      hTxx_rhs = (c11*ztx+c16*zty+c15*ztz)*DzVx + (c16*ztx+c12*zty+c14*ztz)*DzVy + (c15*ztx+c14*zty+c13*ztz)*DzVz; 
      hTyy_rhs = (c12*ztx+c26*zty+c25*ztz)*DzVx + (c26*ztx+c22*zty+c24*ztz)*DzVy + (c25*ztx+c24*zty+c23*ztz)*DzVz;
      hTzz_rhs = (c13*ztx+c36*zty+c35*ztz)*DzVx + (c36*ztx+c23*zty+c34*ztz)*DzVy + (c35*ztx+c34*zty+c33*ztz)*DzVz;
      hTyz_rhs = (c14*ztx+c46*zty+c45*ztz)*DzVx + (c46*ztx+c24*zty+c44*ztz)*DzVy + (c45*ztx+c44*zty+c34*ztz)*DzVz;
      hTxz_rhs = (c15*ztx+c56*zty+c55*ztz)*DzVx + (c56*ztx+c25*zty+c45*ztz)*DzVy + (c55*ztx+c45*zty+c35*ztz)*DzVz;
      hTxy_rhs = (c16*ztx+c66*zty+c56*ztz)*DzVx + (c66*ztx+c26*zty+c46*ztz)*DzVy + (c56*ztx+c46*zty+c36*ztz)*DzVz;

      // 1: make corr to moment equation
      hVx[iptr] += coef_B_minus_1 * hVx_rhs - coef_B * pml_Vx[iptr_a];
      hVy[iptr] += coef_B_minus_1 * hVy_rhs - coef_B * pml_Vy[iptr_a];
      hVz[iptr] += coef_B_minus_1 * hVz_rhs - coef_B * pml_Vz[iptr_a];

      // make corr to Hooke's equatoin
      hTxx[iptr] += coef_B_minus_1 * hTxx_rhs - coef_B * pml_Txx[iptr_a];
      hTyy[iptr] += coef_B_minus_1 * hTyy_rhs - coef_B * pml_Tyy[iptr_a];
      hTzz[iptr] += coef_B_minus_1 * hTzz_rhs - coef_B * pml_Tzz[iptr_a];
      hTxz[iptr] += coef_B_minus_1 * hTxz_rhs - coef_B * pml_Txz[iptr_a];
      hTyz[iptr] += coef_B_minus_1 * hTyz_rhs - coef_B * pml_Tyz[iptr_a];
      hTxy[iptr] += coef_B_minus_1 * hTxy_rhs - coef_B * pml_Txy[iptr_a];
      
      // 2: aux var
      //   a1 = alpha + d / beta, dealt in abs_set_cfspml
      pml_hVx[iptr_a]  = coef_D * hVx_rhs  - coef_A * pml_Vx[iptr_a];
      pml_hVy[iptr_a]  = coef_D * hVy_rhs  - coef_A * pml_Vy[iptr_a];
      pml_hVz[iptr_a]  = coef_D * hVz_rhs  - coef_A * pml_Vz[iptr_a];
      pml_hTxx[iptr_a] = coef_D * hTxx_rhs - coef_A * pml_Txx[iptr_a];
      pml_hTyy[iptr_a] = coef_D * hTyy_rhs - coef_A * pml_Tyy[iptr_a];
      pml_hTzz[iptr_a] = coef_D * hTzz_rhs - coef_A * pml_Tzz[iptr_a];
      pml_hTxz[iptr_a] = coef_D * hTxz_rhs - coef_A * pml_Txz[iptr_a];
      pml_hTyz[iptr_a] = coef_D * hTyz_rhs - coef_A * pml_Tyz[iptr_a];
      pml_hTxy[iptr_a] = coef_D * hTxy_rhs - coef_A * pml_Txy[iptr_a];
    } 
  } 

  return;
}

/*******************************************************************************
 * free surface coef
 * converted matrix for velocity gradient
 *  only implement z2 (top) right now
 ******************************************************************************/

__global__ void
sv_eq1st_curv_col_el_aniso_dvh2dvz_gpu(gdinfo_t        gdinfo_d,
                                     gdcurv_metric_t metric_d,
                                     md_t       md_d,
                                     bdryfree_t      bdryfree_d,
                                     const int verbose)
{
  int ni1 = gdinfo_d.ni1;
  int ni2 = gdinfo_d.ni2;
  int nj1 = gdinfo_d.nj1;
  int nj2 = gdinfo_d.nj2;
  int nk1 = gdinfo_d.nk1;
  int nk2 = gdinfo_d.nk2;
  int nx  = gdinfo_d.nx;
  int ny  = gdinfo_d.ny;
  int nz  = gdinfo_d.nz;
  size_t siz_line   = gdinfo_d.siz_iy;
  size_t siz_slice  = gdinfo_d.siz_iz;
  size_t siz_volume = gdinfo_d.siz_icmp;

  // point to each var
  float * xi_x = metric_d.xi_x;
  float * xi_y = metric_d.xi_y;
  float * xi_z = metric_d.xi_z;
  float * et_x = metric_d.eta_x;
  float * et_y = metric_d.eta_y;
  float * et_z = metric_d.eta_z;
  float * zt_x = metric_d.zeta_x;
  float * zt_y = metric_d.zeta_y;
  float * zt_z = metric_d.zeta_z;

  float * c11d = md_d.c11;
  float * c12d = md_d.c12;
  float * c13d = md_d.c13;
  float * c14d = md_d.c14;
  float * c15d = md_d.c15;
  float * c16d = md_d.c16;
  float * c22d = md_d.c22;
  float * c23d = md_d.c23;
  float * c24d = md_d.c24;
  float * c25d = md_d.c25;
  float * c26d = md_d.c26;
  float * c33d = md_d.c33;
  float * c34d = md_d.c34;
  float * c35d = md_d.c35;
  float * c36d = md_d.c36;
  float * c44d = md_d.c44;
  float * c45d = md_d.c45;
  float * c46d = md_d.c46;
  float * c55d = md_d.c55;
  float * c56d = md_d.c56;
  float * c66d = md_d.c66;

  float *matVx2Vz = bdryfree_d.matVx2Vz2;
  float *matVy2Vz = bdryfree_d.matVy2Vz2;

  float A[3][3], B[3][3], C[3][3];
  float AB[3][3], AC[3][3];

  float c11,c12,c13,c14,c15,c16;
  float     c22,c23,c24,c25,c26;
  float         c33,c34,c35,c36;
  float             c44,c45,c46;
  float                 c55,c56;
  float                     c66;
  float xix, xiy ,xiz, etx, ety, etz, ztx, zty, ztz;
 
  int k = nk2;

  size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  if(ix<(ni2-ni1+1) && iy<(nj2-nj1+1))
  {
    size_t iptr = (ix+ni1) + (iy+nj1) * siz_line + k * siz_slice;

    xix = xi_x[iptr];
    xiy = xi_y[iptr];
    xiz = xi_z[iptr];
    etx = et_x[iptr];
    ety = et_y[iptr];
    etz = et_z[iptr];
    ztx = zt_x[iptr];
    zty = zt_y[iptr];
    ztz = zt_z[iptr];
    
    c11 = c11d[iptr];
    c12 = c12d[iptr];
    c13 = c13d[iptr];
    c14 = c14d[iptr];
    c15 = c15d[iptr];
    c16 = c16d[iptr];
    c22 = c22d[iptr];
    c23 = c23d[iptr];
    c24 = c24d[iptr];
    c25 = c25d[iptr];
    c26 = c26d[iptr];
    c33 = c33d[iptr];
    c34 = c34d[iptr];
    c35 = c35d[iptr];
    c36 = c36d[iptr];
    c44 = c44d[iptr];
    c45 = c45d[iptr];
    c46 = c46d[iptr];
    c55 = c55d[iptr];
    c56 = c56d[iptr];
    c66 = c66d[iptr];

    // first dim: irow; sec dim: jcol, as Fortran code
    A[0][0] = (c11*ztx+c16*zty+c15*ztz)*ztx + (c16*ztx+c66*zty+c56*ztz)*zty + (c15*ztx+c56*zty+c55*ztz)*ztz;
    A[0][1] = (c16*ztx+c12*zty+c14*ztz)*ztx + (c66*ztx+c26*zty+c46*ztz)*zty + (c56*ztx+c25*zty+c45*ztz)*ztz;
    A[0][2] = (c15*ztx+c14*zty+c13*ztz)*ztx + (c56*ztx+c46*zty+c36*ztz)*zty + (c55*ztx+c45*zty+c35*ztz)*ztz; 
    A[1][0] = (c16*ztx+c66*zty+c56*ztz)*ztx + (c12*ztx+c26*zty+c25*ztz)*zty + (c14*ztx+c46*zty+c45*ztz)*ztz; 
    A[1][1] = (c66*ztx+c26*zty+c46*ztz)*ztx + (c26*ztx+c22*zty+c24*ztz)*zty + (c46*ztx+c24*zty+c44*ztz)*ztz; 
    A[1][2] = (c56*ztx+c46*zty+c36*ztz)*ztx + (c25*ztx+c24*zty+c23*ztz)*zty + (c45*ztx+c44*zty+c34*ztz)*ztz;
    A[2][0] = (c15*ztx+c56*zty+c55*ztz)*ztx + (c14*ztx+c46*zty+c45*ztz)*zty + (c13*ztx+c36*zty+c35*ztz)*ztz;
    A[2][1] = (c56*ztx+c25*zty+c45*ztz)*ztx + (c46*ztx+c24*zty+c44*ztz)*zty + (c36*ztx+c23*zty+c34*ztz)*ztz;
    A[2][2] = (c55*ztx+c45*zty+c35*ztz)*ztx + (c45*ztx+c44*zty+c34*ztz)*zty + (c35*ztx+c34*zty+c33*ztz)*ztz; 
    fdlib_math_invert3x3(A);
                                                     
    B[0][0] = (c11*xix+c16*xiy+c15*xiz)*ztx + (c16*xix+c66*xiy+c56*xiz)*zty + (c15*xix+c56*xiy+c55*xiz)*ztz;
    B[0][1] = (c16*xix+c12*xiy+c14*xiz)*ztx + (c66*xix+c26*xiy+c46*xiz)*zty + (c56*xix+c25*xiy+c45*xiz)*ztz;
    B[0][2] = (c15*xix+c14*xiy+c13*xiz)*ztx + (c56*xix+c46*xiy+c36*xiz)*zty + (c55*xix+c45*xiy+c35*xiz)*ztz; 
    B[1][0] = (c16*xix+c66*xiy+c56*xiz)*ztx + (c12*xix+c26*xiy+c25*xiz)*zty + (c14*xix+c46*xiy+c45*xiz)*ztz; 
    B[1][1] = (c66*xix+c26*xiy+c46*xiz)*ztx + (c26*xix+c22*xiy+c24*xiz)*zty + (c46*xix+c24*xiy+c44*xiz)*ztz; 
    B[1][2] = (c56*xix+c46*xiy+c36*xiz)*ztx + (c25*xix+c24*xiy+c23*xiz)*zty + (c45*xix+c44*xiy+c34*xiz)*ztz;
    B[2][0] = (c15*xix+c56*xiy+c55*xiz)*ztx + (c14*xix+c46*xiy+c45*xiz)*zty + (c13*xix+c36*xiy+c35*xiz)*ztz;
    B[2][1] = (c56*xix+c25*xiy+c45*xiz)*ztx + (c46*xix+c24*xiy+c44*xiz)*zty + (c36*xix+c23*xiy+c34*xiz)*ztz;
    B[2][2] = (c55*xix+c45*xiy+c35*xiz)*ztx + (c45*xix+c44*xiy+c34*xiz)*zty + (c35*xix+c34*xiy+c33*xiz)*ztz; 
     
    C[0][0] = (c11*etx+c16*ety+c15*etz)*ztx + (c16*etx+c66*ety+c56*etz)*zty + (c15*etx+c56*ety+c55*etz)*ztz;
    C[0][1] = (c16*etx+c12*ety+c14*etz)*ztx + (c66*etx+c26*ety+c46*etz)*zty + (c56*etx+c25*ety+c45*etz)*ztz;
    C[0][2] = (c15*etx+c14*ety+c13*etz)*ztx + (c56*etx+c46*ety+c36*etz)*zty + (c55*etx+c45*ety+c35*etz)*ztz; 
    C[1][0] = (c16*etx+c66*ety+c56*etz)*ztx + (c12*etx+c26*ety+c25*etz)*zty + (c14*etx+c46*ety+c45*etz)*ztz; 
    C[1][1] = (c66*etx+c26*ety+c46*etz)*ztx + (c26*etx+c22*ety+c24*etz)*zty + (c46*etx+c24*ety+c44*etz)*ztz; 
    C[1][2] = (c56*etx+c46*ety+c36*etz)*ztx + (c25*etx+c24*ety+c23*etz)*zty + (c45*etx+c44*ety+c34*etz)*ztz;
    C[2][0] = (c15*etx+c56*ety+c55*etz)*ztx + (c14*etx+c46*ety+c45*etz)*zty + (c13*etx+c36*ety+c35*etz)*ztz;
    C[2][1] = (c56*etx+c25*ety+c45*etz)*ztx + (c46*etx+c24*ety+c44*etz)*zty + (c36*etx+c23*ety+c34*etz)*ztz;
    C[2][2] = (c55*etx+c45*ety+c35*etz)*ztx + (c45*etx+c44*ety+c34*etz)*zty + (c35*etx+c34*ety+c33*etz)*ztz; 
    fdlib_math_matmul3x3(A, B, AB);
    fdlib_math_matmul3x3(A, C, AC);

    size_t ij = ((iy+nj1) * siz_line + (ix+ni1)) * 9;

    // save into mat
    for(int irow = 0; irow < 3; irow++){
      for(int jcol = 0; jcol < 3; jcol++){
        matVx2Vz[ij + irow*3 + jcol] = -1.0f * AB[irow][jcol];
        matVy2Vz[ij + irow*3 + jcol] = -1.0f * AC[irow][jcol];
      }
    }
  }

  return;
}
